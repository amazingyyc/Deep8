#include "GPUMemoryAllocator.h"

namespace Deep8 {

#ifdef HAVE_CUDA

GPUMemoryAllocator::GPUMemoryAllocator(int deviceId) : deviceId(deviceId) {
}

void* GPUMemoryAllocator::malloc(size_t size) {
	void *ptr;

	CUDA_CHECK(hipSetDevice(deviceId));
	CUDA_CHECK(hipMalloc(&ptr, size));

	return ptr;
}

void GPUMemoryAllocator::free(void *ptr) {
	CUDA_CHECK(hipSetDevice(deviceId));
	CUDA_CHECK(hipFree(ptr));
}

void GPUMemoryAllocator::zero(void *ptr, size_t size) {
	CUDA_CHECK(hipSetDevice(deviceId));
	CUDA_CHECK(hipMemset(ptr, 0, size));
}

/**
 * for GPU the copy function is between the GPU Device
 */
void GPUMemoryAllocator::copy(const void *from, void *to, size_t size) {
	CUDA_CHECK(hipSetDevice(deviceId));
	CUDA_CHECK(hipMemcpy(to, from, size, hipMemcpyDeviceToDevice));
}

/**
 * copy memory from host to GPU
 */
void GPUMemoryAllocator::copyFromCPUToGPU(const void *from, void *to, size_t size) {
	CUDA_CHECK(hipSetDevice(deviceId));
	CUDA_CHECK(hipMemcpy(to, from, size, hipMemcpyHostToDevice));
}

/**
 * copy memory from GPU to Host
 */
void GPUMemoryAllocator::copyFromGPUToCPU(const void *from, void *to, size_t size) {
	CUDA_CHECK(hipSetDevice(deviceId));
	CUDA_CHECK(hipMemcpy(to, from, size, hipMemcpyDeviceToHost));
}

void GPUMemoryAllocator::copyFromGPUToGPU(const void *from, void *to, size_t size) {
	CUDA_CHECK(hipSetDevice(deviceId));
	CUDA_CHECK(hipMemcpy(to, from, size, hipMemcpyDeviceToDevice));
}

#endif

}