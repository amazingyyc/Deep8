#include "hip/hip_runtime.h"
#include "basic/GPUBasic.h"
#include "model/GPUDevice.h"
#include "math/GPUMath.h"
#include "math/Conv2d.h"

namespace Deep8 {
namespace Math {

template <typename T>
__global__ void Conv2dIm2ColKernel(const T* im, 
                                   T* col,
                                   const int batch, 
                                   const int inputHeight, 
                                   const int inputWidth, 
                                   const int inputChannel,
                                   const int outputHeight,
                                   const int outputWidth,
                                   const int filterHeight, 
                                   const int filterWidth,
                                   const int strideY, 
                                   const int strideX, 
                                   const int dilationY, 
                                   const int dilationX,
                                   const int padTop,
                                   const int padLeft,
                                   const int N) {
    int start  = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = start; i < N; i += stride) {
        int colY = i / (filterHeight * filterWidth);
        int colX = i % (filterHeight * filterWidth);

        int b = colY / (outputHeight * outputWidth);

        int outputIndex = colY % (outputHeight * outputWidth);

        int outputY = outputIndex / outputWidth;
        int outputX = outputIndex % outputWidth;

        int filterY = colX / filterWidth;
        int filterX = colX % filterWidth;

        int inputY = padTop + outputY * strideY + filterY * dilationY;
        int inputX = padLeft + outputX * strideX + filterX * dilationX;

        T* colPtr = col + colY * filterHeight * filterWidth * inputChannel + colX * inputChannel;

        if (0 > inputY || inputY >= inputHeight || 0 > inputX || inputX >= inputWidth) {
            for (int k = 0; k < inputChannel; ++k) {
                colPtr[k] = 0;
            }
        } else {
            const T* imPtr = im + b * inputHeight * inputWidth * inputChannel + inputY * inputWidth * inputChannel + inputX * inputChannel;

            for (int k = 0; k < inputChannel; ++k) {
                colPtr[k] = imPtr[k];
            }
        }
    }
}

template <typename T>
__global__ void Conv2dCol2ImKernel(const T* col, 
                                   T* im,
                                   const int batch,
                                   const int inputHeight,
                                   const int inputWidth,
                                   const int inputChannel,
                                   const int outputHeight,
                                   const int outputWidth,
                                   const int filterHeight, 
                                   const int filterWidth, 
                                   const int strideY, 
                                   const int strideX, 
                                   const int dilationY, 
                                   const int dilationX,
                                   const int padTop,
                                   const int padLeft,
                                   const int N) {

    int start  = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    int colWidth = filterHeight * filterWidth * inputChannel;

    for (int i = start; i < N; i += stride) {
        int b = i / (inputHeight * inputWidth);

        int inputIndex = i % (inputHeight * inputWidth);

        int inputY = inputIndex / inputWidth;
        int inputX = inputIndex % inputWidth;

        T* imPtr = im + b * inputHeight * inputWidth * inputChannel + inputY * inputWidth * inputChannel + inputX * inputChannel;

        for (int filterY = 0; filterY < filterHeight; ++filterY) {
            for (int filterX = 0; filterX < filterWidth; ++filterX) {
                int outputY = inputY - padTop - filterY * dilationY;
                int outputX = inputX - padLeft - filterX * dilationX;

                if (0 == (outputY % strideY) && 0 == (outputX % strideX)) {
                    outputY /= strideY;
                    outputX /= strideX;

                    if (0 <= outputY && outputY < outputHeight && 0 <= outputX && outputX < outputWidth) {
                        const T* colPtr = col + (b * outputHeight * outputWidth + outputY * outputWidth + outputX) * colWidth
                            + (filterY * filterWidth + filterX) * inputChannel;

                        for (int k = 0; k < inputChannel; ++k) {
                            imPtr[k] += colPtr[k];
                        }
                    }
                }
            }
        }
    }
}

template <typename T>
void Conv2dGPUImpl(GPUDevice* device,
                   const T *x,
                   const T *y,
                   T *z,
                   T *xcol,
                   const int batch,
                   const int inputHeight,
                   const int inputWidth,
                   const int inputChannel,
                   const int outputHeight,
                   const int outputWidth,
                   const int outputChannel,
                   const int filterHeight,
                   const int filterWidth,
                   const int strideY,
                   const int strideX,
                   const int dilationY,
                   const int dilationX,
                   const int padTop,
                   const int padLeft) {
    DEEP8_RUNTIME_ERROR("the type is error!");
}

template <>
void Conv2dGPUImpl<float>(GPUDevice* device,
                          const float* x,
                          const float* y,
                          float* z,
                          float* xcol,
                          const int batch,
                          const int inputHeight,
                          const int inputWidth,
                          const int inputChannel,
                          const int outputHeight,
                          const int outputWidth,
                          const int outputChannel,
                          const int filterHeight,
                          const int filterWidth,
                          const int strideY,
                          const int strideX,
                          const int dilationY,
                          const int dilationX,
                          const int padTop,
                          const int padLeft) {
    int size = batch * outputHeight * outputWidth * filterHeight * filterWidth;

    int blockSize = DEEP8_GPU_BLOCK_SIZE;
    int grideSize = (size + DEEP8_GPU_BLOCK_SIZE - 1) / DEEP8_GPU_BLOCK_SIZE;

    Conv2dIm2ColKernel<float> << <grideSize, blockSize >> > (x, 
                                                             xcol,
                                                             batch, 
                                                             inputHeight, 
                                                             inputWidth, 
                                                             inputChannel, 
                                                             outputHeight,
                                                             outputWidth,
                                                             filterHeight, 
                                                             filterWidth, 
                                                             padTop, 
                                                             padLeft,
                                                             strideY, 
                                                             strideX, 
                                                             dilationY,
                                                             dilationX,
                                                             size);

    int m = batch * outputHeight * outputWidth;
    int k = filterHeight * filterWidth * inputChannel;
    int n = outputChannel;

    float alpha = 1;
    float beta  = 0;

    CUBLAS_CHECK(hipblasSgemm(device->cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_N, n, m, k, &alpha, y, k, xcol, k, &beta, z, n));
}

template <>
void Conv2dGPUImpl<double>(GPUDevice* device,
                           const double* x,
                           const double* y,
                           double* z,
                           double* xcol,
                           const int batch,
                           const int inputHeight,
                           const int inputWidth,
                           const int inputChannel,
                           const int outputHeight,
                           const int outputWidth,
                           const int outputChannel,
                           const int filterHeight,
                           const int filterWidth,
                           const int strideY,
                           const int strideX,
                           const int dilationY,
                           const int dilationX,
                           const int padTop,
                           const int padLeft) {
    int size = batch * outputHeight * outputWidth * filterHeight * filterWidth;

    int blockSize = DEEP8_GPU_BLOCK_SIZE;
    int grideSize = (size + DEEP8_GPU_BLOCK_SIZE - 1) / DEEP8_GPU_BLOCK_SIZE;

    Conv2dIm2ColKernel<double> << <grideSize, blockSize >> > (x,
                                                              xcol,
                                                              batch,
                                                              inputHeight,
                                                              inputWidth,
                                                              inputChannel,
                                                              outputHeight,
                                                              outputWidth,
                                                              filterHeight,
                                                              filterWidth,
                                                              padTop,
                                                              padLeft,
                                                              strideY,
                                                              strideX,
                                                              dilationY,
                                                              dilationX,
                                                              size);

    int m = batch * outputHeight * outputWidth;
    int k = filterHeight * filterWidth * inputChannel;
    int n = outputChannel;

    double alpha = 1;
    double beta  = 0;

    CUBLAS_CHECK(hipblasDgemm(device->cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_N, n, m, k, &alpha, y, k, xcol, k, &beta, z, n));
}

#ifdef HAVE_HALF
template <>
void Conv2dGPUImpl<half>(GPUDevice* device,
                         const half *x,
                         const half *y,
                         half *z,
                         half *xcol,
                         const int batch,
                         const int inputHeight,
                         const int inputWidth,
                         const int inputChannel,
                         const int outputHeight,
                         const int outputWidth,
                         const int outputChannel,
                         const int filterHeight,
                         const int filterWidth,
                         const int strideY,
                         const int strideX,
                         const int dilationY,
                         const int dilationX,
                         const int padTop,
                         const int padLeft) {
    int size = batch * outputHeight * outputWidth * filterHeight * filterWidth;

    int blockSize = DEEP8_GPU_BLOCK_SIZE;
    int grideSize = (size + DEEP8_GPU_BLOCK_SIZE - 1) / DEEP8_GPU_BLOCK_SIZE;

    Conv2dIm2ColKernel<half> << <grideSize, blockSize >> > (x,
                                                            xcol,
                                                            batch,
                                                            inputHeight,
                                                            inputWidth,
                                                            inputChannel,
                                                            outputHeight,
                                                            outputWidth,
                                                            filterHeight,
                                                            filterWidth,
                                                            padTop,
                                                            padLeft,
                                                            strideY,
                                                            strideX,
                                                            dilationY,
                                                            dilationX,
                                                            size);

    int m = batch * outputHeight * outputWidth;
    int k = filterHeight * filterWidth * inputChannel;
    int n = outputChannel;

    half alpha(1.0);
    half beta(0.0);

    CUBLAS_CHECK(hipblasHgemm(device->cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_N, n, m, k, &alpha, y, k, xcol, k, &beta, z, n));
}
#endif

void Conv2dGPU( const Tensor &x, 
                const Tensor &y, 
                Tensor &z,
                void *xcol,
                bool convered,
                int strideY,
                int strideX,
                int dilationY,
                int dilationX) {
    auto device = (GPUDevice*) x.device();

    auto batch = (int)x.batch();

    auto inputHeight  = (int)x.dim(0);
    auto inputWidth   = (int)x.dim(1);
    auto inputChannel = (int)x.dim(2);

    auto outputHeight  = (int)z.dim(0);
    auto outputWidth   = (int)z.dim(1);
    auto outputChannel = (int)z.dim(2);

    auto filterHeight = (int)y.dim(1);
    auto filterWidth  = (int)y.dim(2);

    auto realFilterHeight = filterHeight + (filterHeight - 1) * (dilationY - 1);
    auto realFilterWidth  = filterWidth  + (filterWidth  - 1) * (dilationX - 1);

    auto padTop  = -(std::max<int>(0, (outputHeight - 1) * strideY + realFilterHeight - inputHeight) / 2);
    auto padLeft = -(std::max<int>(0, (outputWidth  - 1) * strideX + realFilterWidth  - inputWidth)  / 2);

    switch (x.elementType.id) {
    case DType::Float32:
        Conv2dGPUImpl<float>(device,
                          x.data<float>(),
                          y.data<float>(),
                          z.data<float>(),
                          (float*)xcol,
                          batch,
                          inputHeight,
                          inputWidth,
                          inputChannel,
                          outputHeight,
                          outputWidth,
                          outputChannel,
                          filterHeight,
                          filterWidth,
                          strideY,
                          strideX,
                          dilationY,
                          dilationX,
                          padTop,
                          padLeft);
        break;
    case DType::Float64:
        Conv2dGPUImpl<double>(device,
                          x.data<double>(),
                          y.data<double>(),
                          z.data<double>(),
                          (double*)xcol,
                          batch,
                          inputHeight,
                          inputWidth,
                          inputChannel,
                          outputHeight,
                          outputWidth,
                          outputChannel,
                          filterHeight,
                          filterWidth,
                          strideY,
                          strideX,
                          dilationY,
                          dilationX,
                          padTop,
                          padLeft);
        break;
#ifdef HAVE_HALF
    case DType::Float16:
        Conv2dGPUImpl<half>(device,
                          x.data<half>(),
                          y.data<half>(),
                          z.data<half>(),
                          (half*)xcol,
                          batch,
                          inputHeight,
                          inputWidth,
                          inputChannel,
                          outputHeight,
                          outputWidth,
                          outputChannel,
                          filterHeight,
                          filterWidth,
                          strideY,
                          strideX,
                          dilationY,
                          dilationX,
                          padTop,
                          padLeft);
    break;
#endif
    default:
        DEEP8_RUNTIME_ERROR("type " << x.elementType.name << " is not support");
        break;
    }
}

/**grad of x*/
template <typename T>
void Conv2dGradXGPUImpl(GPUDevice* device,
                        const T* x, T* dx,
                        const T* y,
                        const T* z, const T* dz,
                        T *dxcol,
                        const int batch,
                        const int inputHeight,
                        const int inputWidth,
                        const int inputChannel,
                        const int outputHeight,
                        const int outputWidth,
                        const int outputChannel,
                        const int filterHeight,
                        const int filterWidth,
                        const bool convered,
                        const int strideY,
                        const int strideX,
                        const int dilationY,
                        const int dilationX,
                        const int padTop,
                        const int padLeft) {
    DEEP8_RUNTIME_ERROR("the type is error!");
}

template <>
void Conv2dGradXGPUImpl<float>(GPUDevice* device,
                        const float* x, float* dx,
                        const float* y,
                        const float* z, const float* dz,
                        float* dxcol,
                        const int batch,
                        const int inputHeight,
                        const int inputWidth,
                        const int inputChannel,
                        const int outputHeight,
                        const int outputWidth,
                        const int outputChannel,
                        const int filterHeight,
                        const int filterWidth,
                        const bool convered,
                        const int strideY,
                        const int strideX,
                        const int dilationY,
                        const int dilationX,
                        const int padTop,
                        const int padLeft) {
    int size = batch * inputHeight * inputWidth;

    int blockSize = DEEP8_GPU_BLOCK_SIZE;
    int grideSize = (size + DEEP8_GPU_BLOCK_SIZE - 1) / DEEP8_GPU_BLOCK_SIZE;

    int m = filterHeight * filterWidth * inputChannel;
    int k = outputChannel;
    int n = batch * outputHeight * outputWidth;

    float alpha = 1;
    float beta = 0;

    CUBLAS_CHECK(hipblasSgemm(device->cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, y, m, dz, k, &beta, dxcol, m));

    Conv2dCol2ImKernel<float> << <grideSize, blockSize >> > (dxcol,
                                                             dx,
                                                             batch,
                                                             inputHeight,
                                                             inputWidth,
                                                             inputChannel,
                                                             outputHeight,
                                                             outputWidth,
                                                             filterHeight,
                                                             filterWidth,
                                                             strideY,
                                                             strideX,
                                                             dilationY,
                                                             dilationX,
                                                             padTop,
                                                             padLeft,
                                                             size);
}

template <>
void Conv2dGradXGPUImpl<double>(GPUDevice* device,
                               const double* x, double* dx,
                               const double* y,
                               const double* z, const double* dz,
                               double* dxcol,
                               const int batch,
                               const int inputHeight,
                               const int inputWidth,
                               const int inputChannel,
                               const int outputHeight,
                               const int outputWidth,
                               const int outputChannel,
                               const int filterHeight,
                               const int filterWidth,
                               const bool convered,
                               const int strideY,
                               const int strideX,
                               const int dilationY,
                               const int dilationX,
                               const int padTop,
                               const int padLeft) {
    int size = batch * inputHeight * inputWidth;

    int blockSize = DEEP8_GPU_BLOCK_SIZE;
    int grideSize = (size + DEEP8_GPU_BLOCK_SIZE - 1) / DEEP8_GPU_BLOCK_SIZE;

    int m = filterHeight * filterWidth * inputChannel;
    int k = outputChannel;
    int n = batch * outputHeight * outputWidth;

    double alpha = 1;
    double beta = 0;

    CUBLAS_CHECK(hipblasDgemm(device->cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, y, m, dz, k, &beta, dxcol, m));

    Conv2dCol2ImKernel<double> << <grideSize, blockSize >> > (dxcol,
                                                              dx,
                                                              batch,
                                                              inputHeight,
                                                              inputWidth,
                                                              inputChannel,
                                                              outputHeight,
                                                              outputWidth,
                                                              filterHeight,
                                                              filterWidth,
                                                              strideY,
                                                              strideX,
                                                              dilationY,
                                                              dilationX,
                                                              padTop,
                                                              padLeft,
                                                              size);
}

#ifdef HAVE_HALF
template <>
void Conv2dGradXGPUImpl<half>(GPUDevice* device,
                                const half* x, half* dx,
                                const half* y,
                                const half* z, const half* dz,
                                half* dxcol,
                                const int batch,
                                const int inputHeight,
                                const int inputWidth,
                                const int inputChannel,
                                const int outputHeight,
                                const int outputWidth,
                                const int outputChannel,
                                const int filterHeight,
                                const int filterWidth,
                                const bool convered,
                                const int strideY,
                                const int strideX,
                                const int dilationY,
                                const int dilationX,
                                const int padTop,
                                const int padLeft) {
    int size = batch * inputHeight * inputWidth;

    int blockSize = DEEP8_GPU_BLOCK_SIZE;
    int grideSize = (size + DEEP8_GPU_BLOCK_SIZE - 1) / DEEP8_GPU_BLOCK_SIZE;

    int m = filterHeight * filterWidth * inputChannel;
    int k = outputChannel;
    int n = batch * outputHeight * outputWidth;

    half alpha(1.0);
    half beta(0.0);

    CUBLAS_CHECK(hipblasHgemm(device->cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, y, m, dz, k, &beta, dxcol, m));

    Conv2dCol2ImKernel<half> << <grideSize, blockSize >> > (dxcol,
                                                            dx,
                                                            batch,
                                                            inputHeight,
                                                            inputWidth,
                                                            inputChannel,
                                                            outputHeight,
                                                            outputWidth,
                                                            filterHeight,
                                                            filterWidth,
                                                            strideY,
                                                            strideX,
                                                            dilationY,
                                                            dilationX,
                                                            padTop,
                                                            padLeft,
                                                            size);
}
#endif

/**gradient for x (input)*/
void Conv2dGradXGPU(const Tensor& x, 
                    Tensor& dx,
                    const Tensor& y,
                    const Tensor& z, 
                    const Tensor& dz,
                    void *dxcol,
                    bool convered,
                    int strideY,
                    int strideX,
                    int dilationY,
                    int dilationX) {
    auto device = (GPUDevice*) x.device();

    auto batch = (int)x.batch();

    auto inputHeight  = (int)x.dim(0);
    auto inputWidth   = (int)x.dim(1);
    auto inputChannel = (int)x.dim(2);

    auto outputHeight  = (int)z.dim(0);
    auto outputWidth   = (int)z.dim(1);
    auto outputChannel = (int)z.dim(2);

    auto filterHeight = (int)y.dim(1);
    auto filterWidth  = (int)y.dim(2);

    auto realFilterHeight = filterHeight + (filterHeight - 1) * ((int)dilationY - 1);
    auto realFilterWidth  = filterWidth  + (filterWidth  - 1) * ((int)dilationX - 1);

    auto padTop  = -(std::max<int>(0, (outputHeight - 1) * strideY + realFilterHeight - inputHeight) / 2);
    auto padLeft = -(std::max<int>(0, (outputWidth  - 1) * strideX + realFilterWidth  - inputWidth)  / 2);

    switch (x.elementType.id) {
    case DType::Float32:
        Conv2dGradXGPUImpl<float>(device,
                        x.data<float>(), 
                        dx.data<float>(),
                        y.data<float>(),
                        z.data<float>(), 
                        dz.data<float>(),
                        (float*)dxcol,
                        batch,
                        inputHeight,
                        inputWidth,
                        inputChannel,
                        outputHeight,
                        outputWidth,
                        outputChannel,
                        filterHeight,
                        filterWidth,
                        convered,
                        strideY,
                        strideX,
                        dilationY,
                        dilationX,
                        padTop,
                        padLeft);
        break;
    case DType::Float64:
        Conv2dGradXGPUImpl<double>(device,
                        x.data<double>(), 
                        dx.data<double>(),
                        y.data<double>(),
                        z.data<double>(), 
                        dz.data<double>(),
                        (double*)dxcol,
                        batch,
                        inputHeight,
                        inputWidth,
                        inputChannel,
                        outputHeight,
                        outputWidth,
                        outputChannel,
                        filterHeight,
                        filterWidth,
                        convered,
                        strideY,
                        strideX,
                        dilationY,
                        dilationX,
                        padTop,
                        padLeft);
        break;
#ifdef HAVE_HALF
    case DType::Float16:
        Conv2dGradXGPUImpl<half>(device,
                        x.data<half>(), 
                        dx.data<half>(),
                        y.data<half>(),
                        z.data<half>(), 
                        dz.data<half>(),
                        (half*)dxcol,
                        batch,
                        inputHeight,
                        inputWidth,
                        inputChannel,
                        outputHeight,
                        outputWidth,
                        outputChannel,
                        filterHeight,
                        filterWidth,
                        convered,
                        strideY,
                        strideX,
                        dilationY,
                        dilationX,
                        padTop,
                        padLeft);
    break;
#endif
    default:
        DEEP8_RUNTIME_ERROR("type " << x.elementType.name << " is not support");
        break;
    }
}

/**grad for y (filter)*/
template <typename T>
void Conv2dGradYGPUImpl(GPUDevice* device,
                        const T* x,
                        const T* y, T *dy,
                        const T* z, const T* dz,
                        T* xcol,
                        const int batch,
                        const int inputHeight,
                        const int inputWidth,
                        const int inputChannel,
                        const int outputHeight,
                        const int outputWidth,
                        const int outputChannel,
                        const int filterHeight,
                        const int filterWidth,
                        const bool convered,
                        const int strideY,
                        const int strideX,
                        const int dilationY,
                        const int dilationX,
                        const int padTop,
                        const int padLeft) {
    DEEP8_RUNTIME_ERROR("the type is error!");
}

template <>
void Conv2dGradYGPUImpl<float>(GPUDevice* device,
                        const float* x,
                        const float* y, float* dy,
                        const float* z, const float* dz,
                        float* xcol,
                        const int batch,
                        const int inputHeight,
                        const int inputWidth,
                        const int inputChannel,
                        const int outputHeight,
                        const int outputWidth,
                        const int outputChannel,
                        const int filterHeight,
                        const int filterWidth,
                        const bool convered,
                        const int strideY,
                        const int strideX,
                        const int dilationY,
                        const int dilationX,
                        const int padTop,
                        const int padLeft) {
    int size = batch * outputHeight * outputWidth * filterHeight * filterWidth;

    int blockSize = DEEP8_GPU_BLOCK_SIZE;
    int grideSize = (size + DEEP8_GPU_BLOCK_SIZE - 1) / DEEP8_GPU_BLOCK_SIZE;

    Conv2dIm2ColKernel<float> << <grideSize, blockSize >> > (x,
                                                             xcol,
                                                             batch,
                                                             inputHeight,
                                                             inputWidth,
                                                             inputChannel,
                                                             outputHeight,
                                                             outputWidth,
                                                             filterHeight,
                                                             filterWidth,
                                                             strideY,
                                                             strideX,
                                                             dilationY,
                                                             dilationX,
                                                             padTop,
                                                             padLeft,
                                                             size);

    int m = filterHeight * filterWidth * inputChannel;
    int k = batch * outputHeight * outputWidth;
    int n = outputChannel;

    float alpha = 1;
    float beta = 1;

    CUBLAS_CHECK(hipblasSgemm(device->cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_T, m, n, k, &alpha, xcol, m, dz, n, &beta, dy, m));
}

template <>
void Conv2dGradYGPUImpl<double>(GPUDevice* device,
                                const double* x,
                                const double* y, double* dy,
                                const double* z, const double* dz,
                                double* xcol,
                                const int batch,
                                const int inputHeight,
                                const int inputWidth,
                                const int inputChannel,
                                const int outputHeight,
                                const int outputWidth,
                                const int outputChannel,
                                const int filterHeight,
                                const int filterWidth,
                                const bool convered,
                                const int strideY,
                                const int strideX,
                                const int dilationY,
                                const int dilationX,
                                const int padTop,
                                const int padLeft) {
    int size = batch * outputHeight * outputWidth * filterHeight * filterWidth;

    int blockSize = DEEP8_GPU_BLOCK_SIZE;
    int grideSize = (size + DEEP8_GPU_BLOCK_SIZE - 1) / DEEP8_GPU_BLOCK_SIZE;

    Conv2dIm2ColKernel<double> << <grideSize, blockSize >> > (x,
                                                              xcol,
                                                              batch,
                                                              inputHeight,
                                                              inputWidth,
                                                              inputChannel,
                                                              outputHeight,
                                                              outputWidth,
                                                              filterHeight,
                                                              filterWidth,
                                                              strideY,
                                                              strideX,
                                                              dilationY,
                                                              dilationX,
                                                              padTop,
                                                              padLeft,
                                                              size);

    int m = filterHeight * filterWidth * inputChannel;
    int k = batch * outputHeight * outputWidth;
    int n = outputChannel;

    double alpha = 1;
    double beta = 1;

    CUBLAS_CHECK(hipblasDgemm(device->cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_T, m, n, k, &alpha, xcol, m, dz, n, &beta, dy, m));
}

#ifdef HAVE_HALF
template <>
void Conv2dGradYGPUImpl<half>(GPUDevice* device,
                              const half* x,
                              const half* y, half* dy,
                              const half* z, const half* dz,
                              half* xcol,
                              const int batch,
                              const int inputHeight,
                              const int inputWidth,
                              const int inputChannel,
                              const int outputHeight,
                              const int outputWidth,
                              const int outputChannel,
                              const int filterHeight,
                              const int filterWidth,
                              const bool convered,
                              const int strideY,
                              const int strideX,
                              const int dilationY,
                              const int dilationX,
                              const int padTop,
                              const int padLeft) {
    int size = batch * outputHeight * outputWidth * filterHeight * filterWidth;

    int blockSize = DEEP8_GPU_BLOCK_SIZE;
    int grideSize = (size + DEEP8_GPU_BLOCK_SIZE - 1) / DEEP8_GPU_BLOCK_SIZE;

    Conv2dIm2ColKernel<half> << <grideSize, blockSize >> > (x,
                                                            xcol,
                                                            batch,
                                                            inputHeight,
                                                            inputWidth,
                                                            inputChannel,
                                                            outputHeight,
                                                            outputWidth,
                                                            filterHeight,
                                                            filterWidth,
                                                            strideY,
                                                            strideX,
                                                            dilationY,
                                                            dilationX,
                                                            padTop,
                                                            padLeft,
                                                            size);

    int m = filterHeight * filterWidth * inputChannel;
    int k = batch * outputHeight * outputWidth;
    int n = outputChannel;

    half alpha(1.0);
    half beta(1.0);

    CUBLAS_CHECK(hipblasHgemm(device->cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_T, m, n, k, &alpha, xcol, m, dz, n, &beta, dy, m));
}
#endif


void Conv2dGradYGPU(const Tensor &x,
                    const Tensor &y, 
                    Tensor &dy,
                    const Tensor &z, 
                    const Tensor& dz,
                    void *xcol,
                    bool convered,
                    int strideY,
                    int strideX,
                    int dilationY,
                    int dilationX) {
    auto device = (GPUDevice*) x.device();

    auto batch = (int)x.batch();

    auto inputHeight  = (int)x.dim(0);
    auto inputWidth   = (int)x.dim(1);
    auto inputChannel = (int)x.dim(2);

    auto outputHeight  = (int)z.dim(0);
    auto outputWidth   = (int)z.dim(1);
    auto outputChannel = (int)z.dim(2);

    auto filterHeight = (int)y.dim(1);
    auto filterWidth  = (int)y.dim(2);

    auto realFilterHeight = filterHeight + (filterHeight - 1) * ((int)dilationY - 1);
    auto realFilterWidth  = filterWidth  + (filterWidth  - 1) * ((int)dilationX - 1);

    auto padTop  = -(std::max<int>(0, (outputHeight - 1) * strideY + realFilterHeight - inputHeight) / 2);
    auto padLeft = -(std::max<int>(0, (outputWidth  - 1) * strideX + realFilterWidth  - inputWidth)  / 2);

    switch (x.elementType.id) {
    case DType::Float32:
        Conv2dGradYGPUImpl<float>(device,
                        x.data<float>(),
                        y.data<float>(), 
                        dy.data<float>(),
                        z.data<float>(), 
                        dz.data<float>(),
                        (float*)xcol,
                        batch,
                        inputHeight,
                        inputWidth,
                        inputChannel,
                        outputHeight,
                        outputWidth,
                        outputChannel,
                        filterHeight,
                        filterWidth,
                        convered,
                        strideY,
                        strideX,
                        dilationY,
                        dilationX,
                        padTop,
                        padLeft);
        break;
    case DType::Float64:
        Conv2dGradYGPUImpl<double>(device,
                x.data<double>(),
                y.data<double>(), 
                dy.data<double>(),
                z.data<double>(), 
                dz.data<double>(),
                (double*)xcol,
                batch,
                inputHeight,
                inputWidth,
                inputChannel,
                outputHeight,
                outputWidth,
                outputChannel,
                filterHeight,
                filterWidth,
                convered,
                strideY,
                strideX,
                dilationY,
                dilationX,
                padTop,
                padLeft);
        break;
#ifdef HAVE_HALF
    case DType::Float16:
        Conv2dGradYGPUImpl<half>(device,
                x.data<half>(),
                y.data<half>(), 
                dy.data<half>(),
                z.data<half>(), 
                dz.data<half>(),
                (half*)xcol,
                batch,
                inputHeight,
                inputWidth,
                inputChannel,
                outputHeight,
                outputWidth,
                outputChannel,
                filterHeight,
                filterWidth,
                convered,
                strideY,
                strideX,
                dilationY,
                dilationX,
                padTop,
                padLeft);
    break;
#endif
    default:
        DEEP8_RUNTIME_ERROR("type " << x.elementType.name << " is not support");
        break;
    }
}

}
}