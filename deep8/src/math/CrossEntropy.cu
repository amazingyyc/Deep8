#include "hip/hip_runtime.h"
#include "basic/GPUBasic.h"
#include "model/GPUDevice.h"
#include "math/GPUMath.h"
#include "math/GPUReduce.h"
#include "math/CrossEntropy.h"

namespace Deep8 {
namespace Math {

template <typename T>
struct CrossEntropyKernelOp {
	DEEP8_CUDA_FUNC DEEP8_CUDA_INLINE T step(T ret1, T ret2) {
        return ret1 + ret2;
    }
};

template <int blockSize, typename T>
__global__ void CrossEntropyKernel(const T *x, const T *y, T *z, const int batch, const int size) {
    GPUSharedMemory<T> shareMemory;
    T *shared = shareMemory.pointer();

    int blockId = blockIdx.x;
    int threaId = threadIdx.x;

    int i = threaId;

    shared[threaId] = 0;

    while (i < size) {
        shared[threaId] += (-y[i] * cudaLog(x[i]));

        i += blockSize;
    }

    __syncthreads();

    if (blockSize >= 1024) {
        if (threaId < 512) {
            shared[threaId] += shared[threaId + 512];
        }

        __syncthreads();
    }

    if (blockSize >= 512) {
        if (threaId < 256) {
            shared[threaId] += shared[threaId + 256];
        }

        __syncthreads();
    }

    if (blockSize >= 256) {
        if (threaId < 128) {
            shared[threaId] += shared[threaId + 128];
        }

        __syncthreads();
    }

    if (blockSize >= 128) {
        if (threaId < 64) {
            shared[threaId] += shared[threaId + 64];
        }

        __syncthreads();
    }

    if (threaId < 32) {
        Warp32ReduceStep<T, CrossEntropyKernelOp<T>, blockSize>(shared, threaId, CrossEntropyKernelOp<T>());
    }

    if (0 == threaId) {
        z[blockId] = shared[threaId];
    }
}

template <typename T>
void CrossEntropyGPUImpl(GPUDevice *device, 
                         const T *x, 
                         const Shape &xshape, 
                         const T *y, 
                         const Shape &yshape, 
                         T *z, 
                         const Shape &zshape) {
    auto batch = (int) xshape.batch;
    auto size  = (int) xshape.batchSize();

    int gridSize  = batch;
    int blockSize = DEEP8_GPU_BLOCK_SIZE;

    if (blockSize > size) {
        blockSize = prevPowerOf2(size);
    }

    int sharedSize = sizeof(T) * blockSize;

    if (1024 == blockSize) {
        CrossEntropyKernel<1024, T> << <gridSize, blockSize, sharedSize >> > (x, y, z, batch, size);
    } else if (512 == blockSize) {
        CrossEntropyKernel<512,  T> << <gridSize, blockSize, sharedSize >> > (x, y, z, batch, size);
    } else if (256 == blockSize) {
        CrossEntropyKernel<256,  T> << <gridSize, blockSize, sharedSize >> > (x, y, z, batch, size);
    } else if (128 == blockSize) {
        CrossEntropyKernel<128,  T> << <gridSize, blockSize, sharedSize >> > (x, y, z, batch, size);
    } else if (64 == blockSize) {
        CrossEntropyKernel<64,  T> << <gridSize, blockSize, sharedSize >> > (x, y, z, batch, size);
    } else if (32 == blockSize) {
        CrossEntropyKernel<32,  T> << <gridSize, blockSize, sharedSize >> > (x, y, z, batch, size);
    } else if (16 == blockSize) {
        CrossEntropyKernel<16,  T> << <gridSize, blockSize, sharedSize >> > (x, y, z, batch, size);
    } else if (8 == blockSize) {
        CrossEntropyKernel<8,  T> << <gridSize, blockSize, sharedSize >> > (x, y, z, batch, size);
    } else if (4 == blockSize) {
        CrossEntropyKernel<4,  T> << <gridSize, blockSize, sharedSize >> > (x, y, z, batch, size);
    } else if (2 == blockSize) {
        CrossEntropyKernel<2,  T> << <gridSize, blockSize, sharedSize >> > (x, y, z, batch, size);
    } else if (1 == blockSize) {
        CrossEntropyKernel<1,  T> << <gridSize, blockSize, sharedSize >> > (x, y, z, batch, size);
    } else {
        DEEP8_RUNTIME_ERROR("the block size is error");
	}
}

void CrossEntropyGPU(const Tensor &x, const Tensor &y, Tensor &z) {
    auto device = (GPUDevice*) x.device();

    switch (x.elementType.id) {
    case DType::Float32:
        CrossEntropyGPUImpl<float>(device, 
                                x.data<float>(), 
                                x.shape, 
                                y.data<float>(), 
                                y.shape, 
                                z.data<float>(), 
                                z.shape);
        break;
    case DType::Float64:
        CrossEntropyGPUImpl<double>(device, 
                                x.data<double>(), 
                                x.shape, 
                                y.data<double>(), 
                                y.shape, 
                                z.data<double>(), 
                                z.shape);
        break;

#ifdef HAVE_HALF
    case DType::Float16:
        CrossEntropyGPUImpl<half>(device, 
                                x.data<half>(), 
                                x.shape, 
                                y.data<half>(), 
                                y.shape, 
                                z.data<half>(), 
                                z.shape);
        break;
#endif

    default:
        DEEP8_RUNTIME_ERROR("type " << x.elementType.name << " is not support");
        break;
    }
}

template <typename T>
__global__ void CrossEntropyGradXKernel(const T *x, T *dx, const T *y, const T *dz, const int batch, const int size, const int N) {
    int start  = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;

	for (int i = start; i < N; i += stride) {
        int row = i / size;

		dx[i] -= dz[row] * y[i] / x[i];
	}
}

template <typename T>
void CrossEntropyGradXGPUImpl(GPUDevice *device, 
                              const T *x, 
                              T *dx, 
                              const Shape &xshape, 
                              const T *y, 
                              const Shape &yshape, 
                              const T *z, 
                              const T *dz, 
                              const Shape &zshape) {
    auto batch = (int) xshape.batch;
    auto size  = (int) xshape.batchSize();

    int N = (int) xshape.size();

    int blockSize = DEEP8_GPU_BLOCK_SIZE;
    int grideSize = (N + DEEP8_GPU_BLOCK_SIZE - 1) / DEEP8_GPU_BLOCK_SIZE;
    
    CrossEntropyGradXKernel<T><<<grideSize, blockSize >>>(x, dx, y, dz, batch, size, N);
}

void CrossEntropyGradXGPU(const Tensor &x, Tensor &dx, const Tensor &y, const Tensor &z, const Tensor &dz) {
    auto device = (GPUDevice*) x.device();

    switch (x.elementType.id) {
    case DType::Float32:
        CrossEntropyGradXGPUImpl<float>(device, 
            x.data<float>(), 
            dx.data<float>(), 
            x.shape,
            y.data<float>(),
            y.shape,
            z.data<float>(),
            dz.data<float>(),
            z.shape);
        break;
    case DType::Float64:
        CrossEntropyGradXGPUImpl<double>(device, 
            x.data<double>(), 
            dx.data<double>(), 
            x.shape,
            y.data<double>(),
            y.shape,
            z.data<double>(),
            dz.data<double>(),
            z.shape);
        break;

#ifdef HAVE_HALF
    case DType::Float16:
        CrossEntropyGradXGPUImpl<half>(device, 
            x.data<half>(), 
            dx.data<half>(), 
            x.shape,
            y.data<half>(),
            y.shape,
            z.data<half>(),
            dz.data<half>(),
            z.shape);
        break;
#endif

    default:
        DEEP8_RUNTIME_ERROR("type " << x.elementType.name << " is not support");
        break;
    }
}


template <typename T>
__global__ void CrossEntropyGradYKernel(const T *x, const T *y, T *dy, const T *dz, const int batch, const int size, const int N) {
    int start  = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;

	for (int i = start; i < N; i += stride) {
        int row = i / size;

		dy[i] -= dz[row] * cudaLog(x[i]);
	}
}

template <typename T>
void CrossEntropyGradYGPUImpl(GPUDevice *device, 
                              const T *x, 
                              const Shape &xshape, 
                              const T *y, 
                              T *dy,
                              const Shape &yshape, 
                              const T *z, 
                              const T *dz, 
                              const Shape &zshape) {
    auto batch = (int) yshape.batch;
    auto size  = (int) yshape.batchSize();

    int N = (int) yshape.size();

    int blockSize = DEEP8_GPU_BLOCK_SIZE;
    int grideSize = (N + DEEP8_GPU_BLOCK_SIZE - 1) / DEEP8_GPU_BLOCK_SIZE;

    CrossEntropyGradYKernel<T><<<grideSize, blockSize >>>(x, y, dy, dz, batch, size, N);
}

void CrossEntropyGradYGPU(const Tensor &x, const Tensor &y, Tensor &dy, const Tensor &z, const Tensor &dz) {
    auto device = (GPUDevice*) x.device();

    switch (x.elementType.id) {
    case DType::Float32:
        CrossEntropyGradYGPUImpl<float>(device, 
            x.data<float>(), 
            x.shape,
            y.data<float>(),
            dy.data<float>(),
            y.shape,
            z.data<float>(),
            dz.data<float>(),
            z.shape);
        break;
    case DType::Float64:
        CrossEntropyGradYGPUImpl<double>(device, 
            x.data<double>(), 
            x.shape,
            y.data<double>(),
            dy.data<double>(),
            y.shape,
            z.data<double>(),
            dz.data<double>(),
            z.shape);
        break;

#ifdef HAVE_HALF
    case DType::Float16:
        CrossEntropyGradYGPUImpl<half>(device, 
            x.data<half>(), 
            x.shape,
            y.data<half>(),
            dy.data<half>(),
            y.shape,
            z.data<half>(),
            dz.data<half>(),
            z.shape);
        break;
#endif

    default:
        DEEP8_RUNTIME_ERROR("type " << x.elementType.name << " is not support");
        break;
    }
}

}
}