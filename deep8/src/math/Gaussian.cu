#include "basic/GPUBasic.h"
#include "model/GPUDevice.h"
#include "math/GPUUnaryElementWise.h"
#include "math/GPUMath.h"
#include "math/Gaussian.h"

namespace Deep8 {
namespace Math {

template <typename T>
void GaussianGPUImpl(GPUDevice *device, T *x, int size, T mean, T stddev) {
    DEEP8_RUNTIME_ERROR("the type is not suppport");
}

template <>
void GaussianGPUImpl<float>(GPUDevice *device, float *x, int size, float mean, float stddev) {
    /**if 0 != size % 2, than the hiprandGenerateNormal will get error*/
    if (0 == size % 2) {
        CURAND_CHECK(hiprandGenerateNormal(device->curandGenerator, x, size, mean, stddev));
    } else {
        float last = mean;

        CURAND_CHECK(hiprandGenerateNormal(device->curandGenerator, x, size - 1, mean, stddev));

        device->copyFromCPUToGPU(&last, x + (size - 1), sizeof(float));
    }

}

template <>
void GaussianGPUImpl<double>(GPUDevice *device, double *x, int size, double mean, double stddev) {
    if (0 == size % 2) {
        CURAND_CHECK(hiprandGenerateNormalDouble(device->curandGenerator, x, size, mean, stddev));
    } else {
        double last = mean;

        CURAND_CHECK(hiprandGenerateNormalDouble(device->curandGenerator, x, size - 1, mean, stddev));

        device->copyFromCPUToGPU(&last, x + (size - 1), sizeof(double));
    }

}

void GaussianGPU(Tensor &x, float mean, float stddev) {
    auto device = (GPUDevice*)x.device();

    switch (x.elementType.id) {
    case DType::Float32:
        GaussianGPUImpl<float>(device, x.data<float>(), (int)x.size(), mean, stddev);
        break;
    case DType::Float64:
        GaussianGPUImpl<double>(device, x.data<double>(), (int)x.size(), double(mean), double(stddev));
        break;
    default:
        DEEP8_RUNTIME_ERROR("type " << x.elementType.name << " is not support");
        break;
    }
}

}
}