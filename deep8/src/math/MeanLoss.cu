#include "hip/hip_runtime.h"
#include "basic/GPUBasic.h"
#include "model/GPUDevice.h"
#include "math/GPUMath.h"
#include "math/GPUReduce.h"
#include "math/MeanLoss.h"

namespace Deep8 {
namespace Math {

template <typename T>
struct MeanLossKernelOp {
    T ratio;

    MeanLossKernelOp(T r) : ratio(r) {
    }

    DEEP8_CUDA_FUNC DEEP8_CUDA_INLINE T commense() {
        return T(0);
    }

    DEEP8_CUDA_FUNC DEEP8_CUDA_INLINE T init(T ret, T cur) {
        return ret + cur;
    }

    DEEP8_CUDA_FUNC DEEP8_CUDA_INLINE T step(T ret1, T ret2) {
        return ret1 + ret2;
    }

    DEEP8_CUDA_FUNC DEEP8_CUDA_INLINE T complete(T ret) {
        return ret * ratio;
    }
};

void MeanLossGPU(const Tensor& x, Tensor& y) {
    int size = (int)x.shape.size();

    switch (x.elementType.id) {
    case DType::Float32:
        CallReduceKernel<float, MeanLossKernelOp<float>>(
            x.data<float>(),
            y.data<float>(),
            size,
            MeanLossKernelOp<float>(1.0 / float(size)));
        break;
    case DType::Float64:
        CallReduceKernel<double, MeanLossKernelOp<double>>(
            x.data<double>(),
            y.data<double>(),
            size,
            MeanLossKernelOp<double>(1.0 / double(size)));
        break;

#ifdef HAVE_HALF
    case DType::Float16:
        CallReduceKernel<half, MeanLossKernelOp<half>>(
            x.data<half>(),
            y.data<half>(),
            size,
            MeanLossKernelOp<half>(__float2half(1.0 / float(size))));
        break;
#endif

    default:
        DEEP8_RUNTIME_ERROR("type " << x.elementType.name << " is not support");
        break;
    }
}

template <typename T>
struct MeanLossGradKernelOp {
    T ratio;

    MeanLossGradKernelOp(T r) : ratio(r) {
    }

    DEEP8_CUDA_FUNC DEEP8_CUDA_INLINE T operator()(const T& x, const T& y, const T& dy) {
        return dy * ratio;
    }
};

void MeanLossGradGPU(const Tensor& x, Tensor& dx, const Tensor& y, const Tensor& dy) {
    auto xsize = (int)x.shape.size();

    switch (x.elementType.id) {
    case DType::Float32:
        CallReduceGradKernel<float, MeanLossGradKernelOp<float>>(
            x.data<float>(),
            dx.data<float>(),
            y.data<float>(),
            dy.data<float>(),
            xsize,
            MeanLossGradKernelOp<float>(1.0 / float(xsize)));
        break;
    case DType::Float64:
        CallReduceGradKernel<double, MeanLossGradKernelOp<double>>(
            x.data<double>(),
            dx.data<double>(),
            y.data<double>(),
            dy.data<double>(),
            xsize,
            MeanLossGradKernelOp<double>(1.0 / double(xsize)));
        break;

#ifdef HAVE_HALF
    case DType::Float16:
        CallReduceGradKernel<half, MeanLossGradKernelOp<half>>(
            x.data<half>(),
            dx.data<half>(),
            y.data<half>(),
            dy.data<half>(),
            xsize,
            MeanLossGradKernelOp<half>(__float2half(1.0 / float(xsize))));
        break;
#endif

    default:
        DEEP8_RUNTIME_ERROR("type " << x.elementType.name << " is not support");
        break;
    }
}


}
}