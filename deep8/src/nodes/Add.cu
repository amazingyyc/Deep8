#include "hip/hip_runtime.h"
#include "Exception.h"
#include "GPUException.h"
#include "GPUMathUtils.h"
#include "GPUDevice.h"
#include "Add.h"

namespace  Deep8 {

template <typename real>
__global__ void AddForwardKernel(const real *x, const int *xdims, const int *xstrides,
								 const real *y, const int *ydims, const int *ystrides,
									   real *z, const int *zdims, const int *zstrides, const int N) {
	int start  = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;

	int zIndex[MAX_TENSOR_DIMS];

	for (int i = start; i < N; i += stride) {
		for (int k = 0, index = i; k < MAX_TENSOR_DIMS; ++k) {
			zIndex[k] = index / zstrides[k];
			index %= zstrides[k];
		}

		int xIndex = 0;
		int yIndex = 0;

		for (int k = 0; k < MAX_TENSOR_DIMS; ++k) {
			if (xdims[k] == zdims[k]) {
				xIndex += zIndex[k] * xstrides[k];
			}

			if (ydims[k] == zdims[k]) {
				yIndex += zIndex[k] * ystrides[k];
			}
		}

		z[i] = x[xIndex] + y[yIndex];
	}
}

template <typename real>
__global__ void AddForwardKernel_v2(const real *x, NVShape xShape, const real *y, NVShape yShape, real *z, NVShape zShape, const int N) {

}

template <typename real>
__global__ void AddBackwardKernel(real *inGrad,  const int *inShape,  const int *inDims,
							const real *outGrad, const int *outShape, const int *outDims, const int N) {
	int start  = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;

	int inIndex[MAX_TENSOR_DIMS];
	int outIndex[MAX_TENSOR_DIMS];

	for (int inI = start; inI < N; inI += stride) {
		for (int k = 0, index = inI; k < MAX_TENSOR_DIMS; ++k) {
			inIndex[k]  = index / inDims[k];
			outIndex[k] = inIndex[k];

			index %= inDims[k];
		}

		int j = MAX_TENSOR_DIMS - 1;

		while (j >= 0) {
			if (j == MAX_TENSOR_DIMS - 1) {
				int zI = 0;

				for (int l = 0; l < MAX_TENSOR_DIMS; ++l) {
					zI += outIndex[l] * outDims[l];
				}

				inGrad[inI] += outGrad[zI];
			}

			if (inShape[j] == outShape[j]) {
				j--;
			} else {
				outIndex[j]++;

				if (outIndex[j] >= outShape[j]) {
					j--;
				} else {
					for (int l = j + 1; l < MAX_TENSOR_DIMS; ++l) {
						outIndex[l] = inIndex[l];
					}

					j = MAX_TENSOR_DIMS - 1;
				}
			}
		}
	}
}

template <typename T>
void Add<T>::forwardGPUImpl(const T *x, const int *xdims, const int *xstrides,
                            const T *y, const int *ydims, const int *ystrides,
                                  T *z, const int *zdims, const int *zstrides, const int N) {
    int minGrideSize;
    int blockSize;
    int grideSize;

    CUDA_CHECK(hipOccupancyMaxPotentialBlockSize(&minGrideSize, &blockSize, AddForwardKernel<T>, 0, N));

    grideSize = (N + blockSize - 1) / blockSize;

    AddForwardKernel<T> << <grideSize, blockSize >> > (x, xdims, xstrides, y, ydims, ystrides, z, zdims, zstrides, N);
}

#ifdef HAVE_HALF
template <>
void Add<half>::forwardGPUImpl(const half *x, const int *xdims, const int *xstrides,
                               const half *y, const int *ydims, const int *ystrides,
                                     half *z, const int *zdims, const int *zstrides, const int N) {
    int blockSize = 1024;
    int grideSize;

    grideSize = (N + blockSize - 1) / blockSize;

    AddForwardKernel<half> << <grideSize, blockSize >> > (x, xdims, xstrides, y, ydims, ystrides, z, zdims, zstrides, N);
}
#endif

template <typename T>
void Add<T>::forwardGPU(const std::vector<const Tensor<T>*> &inputs, Tensor<T> *output) {
    auto device = static_cast<GPUDevice*>(output->device());

    auto x = inputs[0];
    auto y = inputs[1];
    auto z = output;

    int xshape[MAX_TENSOR_DIMS];
    int yshape[MAX_TENSOR_DIMS];
    int zshape[MAX_TENSOR_DIMS];

    int xdims[MAX_TENSOR_DIMS];
    int ydims[MAX_TENSOR_DIMS];
    int zdims[MAX_TENSOR_DIMS];

    enlongateShapeToMaxDim(x->shape, xshape);
    enlongateShapeToMaxDim(y->shape, yshape);
    enlongateShapeToMaxDim(z->shape, zshape);

    xdims[MAX_TENSOR_DIMS - 1] = 1;
    ydims[MAX_TENSOR_DIMS - 1] = 1;
    zdims[MAX_TENSOR_DIMS - 1] = 1;

    for (int i = MAX_TENSOR_DIMS - 2; i >= 0; --i) {
        xdims[i] = xdims[i + 1] * xshape[i + 1];
        ydims[i] = ydims[i + 1] * yshape[i + 1];
        zdims[i] = zdims[i + 1] * zshape[i + 1];
    }

    auto cudaPtr = (int*)device->malloc(sizeof(int) * MAX_TENSOR_DIMS * 6);

    int *xshapePtr = cudaPtr;
    int *yshapePtr = xshapePtr + MAX_TENSOR_DIMS;
    int *zshapePtr = yshapePtr + MAX_TENSOR_DIMS;
    int *xdimsPtr  = zshapePtr + MAX_TENSOR_DIMS;
    int *ydimsPtr  = xdimsPtr  + MAX_TENSOR_DIMS;
    int *zdimsPtr  = ydimsPtr  + MAX_TENSOR_DIMS;

    device->copyFromCPUToGPU(xshape, xshapePtr, sizeof(int) * MAX_TENSOR_DIMS);
    device->copyFromCPUToGPU(yshape, yshapePtr, sizeof(int) * MAX_TENSOR_DIMS);
    device->copyFromCPUToGPU(zshape, zshapePtr, sizeof(int) * MAX_TENSOR_DIMS);
    device->copyFromCPUToGPU(xdims, xdimsPtr, sizeof(int) * MAX_TENSOR_DIMS);
    device->copyFromCPUToGPU(ydims, ydimsPtr, sizeof(int) * MAX_TENSOR_DIMS);
    device->copyFromCPUToGPU(zdims, zdimsPtr, sizeof(int) * MAX_TENSOR_DIMS);

    forwardGPUImpl(x->data(), xshapePtr, xdimsPtr, y->data(), yshapePtr, ydimsPtr, z->data(), zshapePtr, zdimsPtr, static_cast<int>(z->shape.size()));

    device->free(cudaPtr);
}

template <typename T>
void Add<T>::backwardGPUImpl(T *inGrad, const int *inShape, const int *inDims, const T *outGrad, const int *outShape, const int *outDims, const int N) {
    int minGrideSize;
    int blockSize;
    int grideSize;

    CUDA_CHECK(hipOccupancyMaxPotentialBlockSize(&minGrideSize, &blockSize, AddBackwardKernel<T>, 0, N));

    grideSize = (N + blockSize - 1) / blockSize;

    AddBackwardKernel<T> << <grideSize, blockSize >> > (inGrad, inShape, inDims, outGrad, outShape, outDims, N);
}

#ifdef HAVE_HALF
template <>
void Add<half>:: backwardGPUImpl(half *inGrad, const int *inShape, const int *inDims, const half *outGrad, const int *outShape, const int *outDims, const int N) {
    int blockSize = 1024;
    int grideSize;

    grideSize = (N + blockSize - 1) / blockSize;

    AddBackwardKernel<half> << <grideSize, blockSize >> > (inGrad, inShape, inDims, outGrad, outShape, outDims, N);
}
#endif

template <typename T>
void Add<T>::backwardGPU(const std::vector<const Tensor<T>*> &inputs, const Tensor<T> *output, const Tensor<T> *outputGradient, size_t index, Tensor<T> *iGradient) {
    DEEP8_ARGUMENT_CHECK(0 == index || 1 == index, "the index is error");

    auto device = static_cast<GPUDevice*>(iGradient->device());

    int inShape[MAX_TENSOR_DIMS];
    int outShape[MAX_TENSOR_DIMS];

    int inDims[MAX_TENSOR_DIMS];
    int outDims[MAX_TENSOR_DIMS];

    enlongateShapeToMaxDim(iGradient->shape, inShape);
    enlongateShapeToMaxDim(outputGradient->shape, outShape);

    inDims[MAX_TENSOR_DIMS - 1] = 1;
    outDims[MAX_TENSOR_DIMS - 1] = 1;

    for (int i = MAX_TENSOR_DIMS - 2; i >= 0; --i) {
        inDims[i]  = inDims[i + 1] * inShape[i + 1];
        outDims[i] = outDims[i + 1] * outShape[i + 1];
    }


    auto cudaPtr = (int*)device->malloc(sizeof(int) * MAX_TENSOR_DIMS * 4);

    int *inShapePtr  = cudaPtr;
    int *outShapePtr = inShapePtr  + MAX_TENSOR_DIMS;
    int *inDimsPtr   = outShapePtr + MAX_TENSOR_DIMS;
    int *outDimsPtr  = inDimsPtr   + MAX_TENSOR_DIMS;

    device->copyFromCPUToGPU(inShape,   inShapePtr, sizeof(int) * MAX_TENSOR_DIMS);
    device->copyFromCPUToGPU(outShape, outShapePtr, sizeof(int) * MAX_TENSOR_DIMS);
    device->copyFromCPUToGPU(inDims,   inDimsPtr, sizeof(int) * MAX_TENSOR_DIMS);
    device->copyFromCPUToGPU(outDims, outDimsPtr, sizeof(int) * MAX_TENSOR_DIMS);

    backwardGPUImpl(iGradient->data(), inShapePtr, inDimsPtr,
                    outputGradient->data(), outShapePtr, outDimsPtr, static_cast<int>(iGradient->size()));

    device->free(cudaPtr);
}

DEEP8_DECLARATION_GPU_FUNC(Add);

template void Add<float>::forwardGPUImpl(const float *x, const int *xdims, const int *xstrides,
										 const float *y, const int *ydims, const int *ystrides,
										       float *z, const int *zdims, const int *zstrides, const int N);
template void Add<double>::forwardGPUImpl(const double *x, const int *xdims, const int *xstrides,
										  const double *y, const int *ydims, const int *ystrides,
												double *z, const int *zdims, const int *zstrides, const int N);

#ifdef HAVE_HALF
template void Add<half>::forwardGPUImpl(const half *x, const int *xdims, const int *xstrides,
										const half *y, const int *ydims, const int *ystrides,
										half *z, const int *zdims, const int *zstrides, const int N);
#endif

template void Add<float>::backwardGPUImpl(float *inGrad, const int *inShape, const int *inDims, const float *outGrad, const int *outShape, const int *outDims, const int N);
template void Add<double>::backwardGPUImpl(double *inGrad, const int *inShape, const int *inDims, const double *outGrad, const int *outShape, const int *outDims, const int N);
#ifdef HAVE_HALF
template void Add<half>::backwardGPUImpl(half *inGrad, const int *inShape, const int *inDims, const half *outGrad, const int *outShape, const int *outDims, const int N);
#endif


}