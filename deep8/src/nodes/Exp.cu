#include "hip/hip_runtime.h"
#include "Exception.h"
#include "GPUException.h"
#include "GPUMathUtils.h"
#include "GPUDevice.h"
#include "GPUElementWise.cuh"
#include "Exp.h"

namespace Deep8 {

template <typename T>
struct ExpForwardOP {
	DEEP8_CUDA_FUNC DEEP8_CUDA_INLINE T forward(const T &x) {}
};

template <>
struct ExpForwardOP<float> {
	DEEP8_CUDA_FUNC DEEP8_CUDA_INLINE float forward(const float &x) {
		return expf(x);
	}
};

template <>
struct ExpForwardOP<double> {
	DEEP8_CUDA_FUNC DEEP8_CUDA_INLINE double forward(const double &x) {
		return exp(x);
	}
};

#ifdef HAVE_HALF
template <>
struct ExpForwardOP<half> {
	DEEP8_CUDA_FUNC DEEP8_CUDA_INLINE half forward(const half &x) {
		return hexp(x);
	}
};
#endif

template <typename T>
struct ExpBackwardOP {
	DEEP8_CUDA_FUNC DEEP8_CUDA_INLINE T backward(const T &x, const T &y, const T &dy) {
		return y * dy;
	}
};

template <typename T>
void Exp<T>::forwardGPU(const std::vector<const Tensor<T>*> &inputs, Tensor<T> *output) {
	auto x = inputs[0]->data();
	auto y = output->data();
	auto N = (int)output->shape.size();

	int grideSize = (N + DEEP8_GPU_BLOCK_SIZE - 1) / DEEP8_GPU_BLOCK_SIZE;

	UnaryElementWiseForward<T, ExpForwardOP<T>> << <grideSize, DEEP8_GPU_BLOCK_SIZE >> > (x, y, ExpForwardOP<T>(), N);
}

template <typename T>
void Exp<T>::backwardGPU(const std::vector<const Tensor<T>*> &inputs,
                         const Tensor<T> *output,
                         const Tensor<T> *outputGradient,
                         size_t index,
                         Tensor<T> *iGradient) {
    DEEP8_ARGUMENT_CHECK(0 == index, "the index of Exp backwardCPU is error");

	auto x  = inputs[0]->data();
	auto dx = iGradient->data();
	auto y  = output->data();
	auto dy = outputGradient->data();

	int N = (int)iGradient->shape.size();

	int grideSize = (N + DEEP8_GPU_BLOCK_SIZE - 1) / DEEP8_GPU_BLOCK_SIZE;

	UnaryElementWiseBackward<T, ExpBackwardOP<T>> << <grideSize, DEEP8_GPU_BLOCK_SIZE >> > (x, dx, y, dy, ExpBackwardOP<T>(), N);
}

DEEP8_DECLARATION_GPU_FUNC(Exp);

}