#include "hip/hip_runtime.h"
#include "Exception.h"
#include "GPUException.h"
#include "GPUMathUtils.h"
#include "GPUDevice.h"
#include "GPUReduce.h"
#include "Softmax.h"

namespace Deep8 {

/**
 * find the max value and put it in y
 */
template <typename T>
struct SoftmaxFindMaxOp {
	DEEP8_CUDA_FUNC DEEP8_CUDA_INLINE T commense() {
		return CuMath::cuMinValue<T>();
	}

	DEEP8_CUDA_FUNC DEEP8_CUDA_INLINE T init(T ret, T cur) {
		return ret >= cur ? ret : cur;
	}

	DEEP8_CUDA_FUNC DEEP8_CUDA_INLINE T step(T ret1, T ret2) {
		return ret1 >= ret2 ? ret1 : ret2;
	}

	DEEP8_CUDA_FUNC DEEP8_CUDA_INLINE T complete(T ret) {
		return ret;
	}
};


/**
 * Y[i] = exp(X[i] - scalar);
 */
template <typename real>
__global__ void SoftmaxExpMinusScalar(const real *x, const real *scalar, real *y, const int size, const int N) {
    int start  = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = start; i < N; i += stride) {
        y[i] = CuMath::cuExp(x[i] - scalar[i / size]);
    }
}

template <typename T>
struct SoftmaxSumOp {
	DEEP8_CUDA_FUNC DEEP8_CUDA_INLINE T commense() {
		return T(0);
	}

	DEEP8_CUDA_FUNC DEEP8_CUDA_INLINE T init(T ret, T cur) {
		return ret + cur;
    }

	DEEP8_CUDA_FUNC DEEP8_CUDA_INLINE T step(T ret1, T ret2) {
        return ret1 + ret2;
    }

	DEEP8_CUDA_FUNC DEEP8_CUDA_INLINE T complete(T ret) {
        return ret;
    }
};

/**
 * Y[i] = X[i] / scalar[0];
 */
template <typename real>
__global__ void SoftmaxDivideScalar(real *y, const real *scalar, const int size, const int N) {
    int start = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = start; i < N; i += stride) {
        y[i] = y[i] / scalar[i / size];
    }
}

template <int blockSize, typename real>
__global__ void SoftmaxBackwardDotKernel(const real *y, const real *dy, real *dotPtr, const int batch, const int size) {
    SharedMemory<real> shareMemory;
    real *shared = shareMemory.pointer();

    int threaId = threadIdx.x;
    int blockId = blockIdx.x;

    int i = blockId * size + threaId;
    int j = threaId;

    shared[threaId] = 0;

    while (j < size) {
        shared[threaId] += y[i] * dy[i];

        i += blockSize;
        j += blockSize;
    }

    __syncthreads();

    if (blockSize >= 1024) {
        if (threaId < 512) {
            shared[threaId] += shared[threaId + 512];
        }

        __syncthreads();
    }

    if (blockSize >= 512) {
        if (threaId < 256) {
            shared[threaId] += shared[threaId + 256];
        }

        __syncthreads();
    }

    if (blockSize >= 256) {
        if (threaId < 128) {
            shared[threaId] += shared[threaId + 128];
        }

        __syncthreads();
    }

    if (blockSize >= 128) {
        if (threaId < 64) {
            shared[threaId] += shared[threaId + 64];
        }

        __syncthreads();
    }

    if (threaId < 32) {
		warpSumReduce<blockSize, real>(shared, threaId);
    }

    if (0 == threaId) {
        dotPtr[blockId] = shared[threaId];
    }
}

template <typename real>
__global__ void SoftmaxBackwardKernel(real *dx, const real *y, const real *dy, const real *scalar, const int size, const int N) {
    int start  = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = start; i < N; i += stride) {
        dx[i] += (dy[i] - scalar[i / size]) * y[i];
    }
}

template <typename T>
void Softmax<T>::forwardGPU(const std::vector<const Tensor<T>*> &inputs, Tensor<T> *output) {
	auto device = (GPUDevice*)output->device();

	auto xshape = inputs[0]->shape;

	auto x = inputs[0]->data();
	auto y = output->data();

	int dim0 = xshape.batch;
	int dim1 = xshape.dim(axis);
	int dim2 = 1;

	for (int i = 0; i < axis; ++i) {
		dim0 *= xshape.dim(i);
	}

	for (int i = axis + 1; i < xshape.nDims; ++i) {
		dim2 *= xshape.dim(i);
	}

	auto maxPtr = (T*)device->malloc(sizeof(T) * dim0 * dim2);
	auto sumPtr = (T*)device->malloc(sizeof(T) * dim0 * dim2);

	/**find max value*/
	if (1 == dim2) {
		/**tail reduce*/

	} else if (1 == dim0) {
		/**head reduce*/

	} else {
		/**middle reduce*/

	}

	/***/

	/*
    auto device = (GPUDevice*)output->device();

    auto x = inputs[0]->data();
    auto y = output->data();

    int N      = (int)output->shape.size();
    int batch  = (int)output->shape.batch;
    int size   = N / batch;

    int blockSize = 1024;

    if (size < blockSize) {
        blockSize = prevPowerOf2(size);
    }

    auto maxPtr = (T*)device->malloc(sizeof(T) * batch);
    auto sumPtr = (T*)device->malloc(sizeof(T) * batch);
	*/


    /**find max*/

	/*
    callTailReduceForward<T, SoftmaxFindMaxOp<T>>(x, maxPtr, batch, size, blockSize);

    int grideSize = (N + DEEP8_GPU_BLOCK_SIZE - 1) / DEEP8_GPU_BLOCK_SIZE;

    SoftmaxExpMinusScalar<T><<<grideSize, DEEP8_GPU_BLOCK_SIZE >>>(x, maxPtr, y, size, N);
	*/

    /***calculate sum*/
    /*
	callTailReduceForward<T, SoftmaxSumOp<T>>(y, sumPtr, batch, size, blockSize);

    SoftmaxDivideScalar<T><<<grideSize, DEEP8_GPU_BLOCK_SIZE >>>(y, sumPtr, size, N);

    device->free(sumPtr);
    device->free(maxPtr);
	*/
}

template <typename T>
void Softmax<T>::backwardGPU(const std::vector<const Tensor<T>*> &inputs, const Tensor<T> *output, const Tensor<T> *outputGradient, size_t index, Tensor<T> *iGradient) {
	/*
    DEEP8_ARGUMENT_CHECK(0 == index, "the index of Softmax backwardCPU is error");

	auto device = (GPUDevice*)iGradient->device();

	auto dx = iGradient->data();
	auto y  = output->data();
	auto dy = outputGradient->data();

    int N      = (int)iGradient->shape.size();
    int batch  = (int)iGradient->shape.batch;
    int size   = N / batch;

    int blockSize = 1024;

    if (size < blockSize) {
        blockSize = prevPowerOf2(size);
    }

    int sharedSize = sizeof(T) * blockSize;

    auto dotPtr = (T*)device->malloc(sizeof(T) * batch);

    if (1024 == blockSize) {
        SoftmaxBackwardDotKernel<1024, T> << <batch, blockSize, sharedSize >> > (y, dy, dotPtr, batch, size);
    } else if (512 == blockSize) {
        SoftmaxBackwardDotKernel<512,  T> << <batch, blockSize, sharedSize >> > (y, dy, dotPtr, batch, size);
    } else if (256 == blockSize) {
        SoftmaxBackwardDotKernel<256,  T> << <batch, blockSize, sharedSize >> > (y, dy, dotPtr, batch, size);
    } else if (128 == blockSize) {
        SoftmaxBackwardDotKernel<128,  T> << <batch, blockSize, sharedSize >> > (y, dy, dotPtr, batch, size);
    } else if (64 == blockSize) {
        SoftmaxBackwardDotKernel<64,  T> << <batch, blockSize, sharedSize >> > (y, dy, dotPtr, batch, size);
    } else if (32 == blockSize) {
        SoftmaxBackwardDotKernel<32,  T> << <batch, blockSize, sharedSize >> > (y, dy, dotPtr, batch, size);
    } else if (16 == blockSize) {
        SoftmaxBackwardDotKernel<16,  T> << <batch, blockSize, sharedSize >> > (y, dy, dotPtr, batch, size);
    } else if (8 == blockSize) {
        SoftmaxBackwardDotKernel<8,  T> << <batch, blockSize, sharedSize >> > (y, dy, dotPtr, batch, size);
    } else if (4 == blockSize) {
        SoftmaxBackwardDotKernel<4,  T> << <batch, blockSize, sharedSize >> > (y, dy, dotPtr, batch, size);
    } else if (2 == blockSize) {
        SoftmaxBackwardDotKernel<2,  T> << <batch, blockSize, sharedSize >> > (y, dy, dotPtr, batch, size);
    } else if (1 == blockSize) {
        SoftmaxBackwardDotKernel<1,  T> << <batch, blockSize, sharedSize >> > (y, dy, dotPtr, batch, size);
    } else {
        DEEP8_RUNTIME_ERROR("the block size is error");
	}

    int grideSize = (N + DEEP8_GPU_BLOCK_SIZE - 1) / DEEP8_GPU_BLOCK_SIZE;

    SoftmaxBackwardKernel<T><<<grideSize, DEEP8_GPU_BLOCK_SIZE >>>(dx, y, dy, dotPtr, size, N);

    device->free(dotPtr);
	*/
}

DEEP8_DECLARATION_GPU_FUNC(Softmax);

}