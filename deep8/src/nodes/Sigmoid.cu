#include "hip/hip_runtime.h"
#include "Exception.h"
#include "GPUException.h"
#include "GPUMathUtils.h"
#include "GPUDevice.h"
#include "GPUElementWise.cuh"
#include "Sigmoid.h"

namespace Deep8 {

template <typename T>
struct SigmoidForwardOP {
	DEEP8_CUDA_FUNC DEEP8_CUDA_INLINE T forward(const T &x) {}
};

template <>
struct SigmoidForwardOP<float> {
	DEEP8_CUDA_FUNC DEEP8_CUDA_INLINE float forward(const float &x) {
		return 0.5 + 0.5 * tanhf(0.5 * x);
	}
};

template <>
struct SigmoidForwardOP<double> {
	DEEP8_CUDA_FUNC DEEP8_CUDA_INLINE double forward(const double &x) {
		return 0.5 + 0.5 * tanh(0.5 * x);
	}
};

#ifdef HAVE_HALF
template <>
struct SigmoidForwardOP<half> {
	DEEP8_CUDA_FUNC DEEP8_CUDA_INLINE half forward(const half &x) {
		return __float2half(0.5 + 0.5 * tanhf(0.5 * __half2float(x)));
	}
};
#endif

template <typename T>
struct SigmoidBackwardOP {
	DEEP8_CUDA_FUNC DEEP8_CUDA_INLINE T backward(const T &x, const T &y, const T &dy) {
		return dy * y * (T(1) - y);
	}
};

//template <typename real>
//__global__ void SigmoidForwardKernel(const real *X, real *Y, const int N) {
//    int start = blockIdx.x * blockDim.x + threadIdx.x;
//    int stride = blockDim.x * gridDim.x;
//
//    for (int i = start; i < N; i += stride) {
//        Y[i] = real(0.5) + real(0.5) * cuTanh(real(0.5) * X[i]);
//    }
//}
//
//template <typename real>
//__global__ void SigmoidBackwardKernel(real *xGrad, const real *yGrad, const real *Y, const int N) {
//    int start = blockIdx.x * blockDim.x + threadIdx.x;
//    int stride = blockDim.x * gridDim.x;
//
//    for (int i = start; i < N; i += stride) {
//        xGrad[i] += yGrad[i] * Y[i] * (real(1) - Y[i]);
//    }
//}

template <typename T>
void Sigmoid<T>::forwardGPU(const std::vector<const Tensor<T>*> &inputs, Tensor<T> *output) {
	auto x = inputs[0]->data();
	auto y = output->data();
	auto N = (int)output->shape.size();

	int grideSize = (N + DEEP8_GPU_BLOCK_SIZE - 1) / DEEP8_GPU_BLOCK_SIZE;

	UnaryElementWiseForward<T, SigmoidForwardOP<T>> << <grideSize, DEEP8_GPU_BLOCK_SIZE >> > (x, y, SigmoidForwardOP<T>(), N);
}

template <typename T>
void Sigmoid<T>::backwardGPU(const std::vector<const Tensor<T>*> &inputs,
                             const Tensor<T> *output,
                             const Tensor<T> *outputGradient,
                             size_t index,
                             Tensor<T> *iGradient) {
	DEEP8_ARGUMENT_CHECK(0 == index, "the index of is error");

	auto x  = inputs[0]->data();
	auto dx = iGradient->data();
	auto y  = output->data();
	auto dy = outputGradient->data();

	int N = (int)iGradient->shape.size();

	int grideSize = (N + DEEP8_GPU_BLOCK_SIZE - 1) / DEEP8_GPU_BLOCK_SIZE;

	UnaryElementWiseBackward<T, SigmoidBackwardOP<T>> << <grideSize, DEEP8_GPU_BLOCK_SIZE >> > (x, dx, y, dy, SigmoidBackwardOP<T>(), N);
}


DEEP8_DECLARATION_GPU_FUNC(Sigmoid);

}