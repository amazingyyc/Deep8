#include "hip/hip_runtime.h"
#include "Trainer.h"
#include "../basic/GPUDevice.h"

namespace Deep8 {

/**********************************************************************/
/**Trainer*/
/**********************************************************************/

#ifdef HAVE_HALF
template <int blockSize>
__global__ void TrainerNorm2HalfKernel(const half *x, float *y, const int size) {
	SharedMemory<float> shareMemory;
	float *shared = shareMemory.pointer();

	int threaId = threadIdx.x;

	int j = threaId;

	shared[threaId] = 0;

	while (j < size) {
		shared[threaId] += __half2float(x[j]) * __half2float(x[j]);

		j += blockSize;
	}

	__syncthreads();

	if (blockSize >= 1024) {
		if (threaId < 512) {
			shared[threaId] += shared[threaId + 512];
		}

		__syncthreads();
	}

	if (blockSize >= 512) {
		if (threaId < 256) {
			shared[threaId] += shared[threaId + 256];
		}

		__syncthreads();
	}

	if (blockSize >= 256) {
		if (threaId < 128) {
			shared[threaId] += shared[threaId + 128];
		}

		__syncthreads();
	}

	if (blockSize >= 128) {
		if (threaId < 64) {
			shared[threaId] += shared[threaId + 64];
		}

		__syncthreads();
	}

	if (threaId < 32) {
		warpSumReduce<blockSize, float>(shared, threaId);
	}

	if (0 == threaId) {
		y[0] = shared[threaId];
	}
}
#endif

template <>
float Trainer<float>::clipGradientScaleGPU(Device *d, std::unordered_set<Parameter<float>*> &parameters, float clipThreshold) {
#ifdef HAVE_CUDA
	auto device = (GPUDevice*)d;

	std::vector<float> l2NormVec;

	for (auto node : parameters) {
		if (!node->updateGradient) {
			continue;
		}

		auto parameter = node;
		auto gradient  = parameter->gradient;

		l2NormVec.push_back(float(0));

		CUBLAS_CHECK(hipblasSnrm2(device->cublasHandle, (int)gradient.size(), gradient.data(), 1, &(l2NormVec[l2NormVec.size() - 1])));
	}

	float sum = 0;

	for (auto item : l2NormVec) {
		sum += item;
	}


	auto scale = clipThreshold / std::sqrt(sum);

	if (isnan(scale) || isinf(scale)) {
		return 1;
	}

	return scale;
#else
	DEEP8_RUNTIME_ERROR("does not have a GPU");
#endif
}

template <>
double Trainer<double>::clipGradientScaleGPU(Device *d, std::unordered_set<Parameter<double>*> &parameters, double clipThreshold) {
#ifdef HAVE_CUDA
	auto device = (GPUDevice*)d;

	std::vector<double> l2NormVec;

	for (auto node : parameters) {
		if (!node->updateGradient) {
			continue;
		}

		auto parameter = node;
		auto gradient = parameter->gradient;

		l2NormVec.push_back(double(0));

		CUBLAS_CHECK(hipblasDnrm2(device->cublasHandle, (int)gradient.size(), gradient.data(), 1, &(l2NormVec[l2NormVec.size() - 1])));
	}

	double sum = 0;

	for (auto item : l2NormVec) {
		sum += item;
	}

	auto scale = clipThreshold / std::sqrt(sum);

	if (isnan(scale) || isinf(scale)) {
		return 1;
	}

	return scale;

#else
DEEP8_RUNTIME_ERROR("does not have a GPU");
#endif
}

#ifdef HAVE_HALF
template <>
half Trainer<half>::clipGradientScaleGPU(Device *d, std::unordered_set<Parameter<half>*> &parameters, half clipThreshold) {
	auto device = (GPUDevice*)d;

	int updateCount = 0;

	for (auto node : parameters) {
		if (node->updateGradient) {
			updateCount++;
		}
	}

	if (0 >= updateCount) {
		return 1.0;
	}

	float *sumPtr = (float*)device->malloc(sizeof(float) * updateCount);

	int index = 0;

	for (auto node : parameters) {
		if (!node->updateGradient) {
			continue;
		}

		auto parameter = node;
		auto gradient = parameter->gradient;

		int size = (int)gradient.size();

		int blockSize = 1024;

		if (size < blockSize) {
			blockSize = prevPowerOf2(size);
		}

		int sharedSize = sizeof(float) * blockSize;

		if (1024 == blockSize) {
			TrainerNorm2HalfKernel<1024> << <1, blockSize, sharedSize >> > (gradient.data(), sumPtr + index, size);
		} else if (512 == blockSize) {
			TrainerNorm2HalfKernel<512> << <1, blockSize, sharedSize >> > (gradient.data(), sumPtr + index, size);
		} else if (256 == blockSize) {
			TrainerNorm2HalfKernel<256> << <1, blockSize, sharedSize >> > (gradient.data(), sumPtr + index, size);
		} else if (128 == blockSize) {
			TrainerNorm2HalfKernel<128> << <1, blockSize, sharedSize >> > (gradient.data(), sumPtr + index, size);
		} else if (64 == blockSize) {
			TrainerNorm2HalfKernel<64> << <1, blockSize, sharedSize >> > (gradient.data(), sumPtr + index, size);
		} else if (32 == blockSize) {
			TrainerNorm2HalfKernel<32> << <1, blockSize, sharedSize >> > (gradient.data(), sumPtr + index, size);
		} else if (16 == blockSize) {
			TrainerNorm2HalfKernel<16> << <1, blockSize, sharedSize >> > (gradient.data(), sumPtr + index, size);
		} else if (8 == blockSize) {
			TrainerNorm2HalfKernel<8> << <1, blockSize, sharedSize >> > (gradient.data(), sumPtr + index, size);
		} else if (4 == blockSize) {
			TrainerNorm2HalfKernel<4> << <1, blockSize, sharedSize >> > (gradient.data(), sumPtr + index, size);
		} else if (2 == blockSize) {
			TrainerNorm2HalfKernel<2> << <1, blockSize, sharedSize >> > (gradient.data(), sumPtr + index, size);
		} else if (1 == blockSize) {
			TrainerNorm2HalfKernel<1> << <1, blockSize, sharedSize >> > (gradient.data(), sumPtr + index, size);
		} else {
			DEEP8_RUNTIME_ERROR("the block size is error");
		}

		index++;
	}

	std::vector<float> l2NormVec(updateCount);

	device->copyFromGPUToCPU(sumPtr, &l2NormVec[0], sizeof(float) * updateCount);
	device->free(sumPtr);

	float sum = 0;

	for (auto item : l2NormVec) {
		sum += item;
	}

	float floatClipThreshold = __half2float(clipThreshold);
	float scale = floatClipThreshold / std::sqrt(sum);

	if (isnan(scale) || isinf(scale)) {
		return 1.0;
	}

	return half(scale);
}
#endif

/**********************************************************************/
/**SGDTrainer*/
/**********************************************************************/
#ifdef HAVE_CUDA
template <typename real>
__global__ void SGDTrainerKernel(real *gradient, const real scale, const real learningRate, real *value, const int N) {
	int start = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;

	for (int i = start; i < N; i += stride) {
		value[i] -= scale * learningRate * gradient[i];
	}
}
#endif

template <>
void SGDTrainer<float>::trainingGPU(Parameter<float> *parameter, float scale) {
	auto value    = parameter->value;
	auto gradient = parameter->gradient;

	auto device = static_cast<GPUDevice*>(value.device());

	float alpha = -1 * (this->learningRate * scale);

	CUBLAS_CHECK(hipblasSaxpy(device->cublasHandle, (int)value.size(), &alpha, gradient.data(), 1, value.data(), 1));
}

template <>
void SGDTrainer<double>::trainingGPU(Parameter<double> *parameter, double scale) {
	auto value = parameter->value;
	auto gradient = parameter->gradient;

	auto device = static_cast<GPUDevice*>(value.device());

	double alpha = -1 * (this->learningRate * scale);

	CUBLAS_CHECK(hipblasDaxpy(device->cublasHandle, (int)value.size(), &alpha, gradient.data(), 1, value.data(), 1));
}

#ifdef HAVE_HALF
template <>
void SGDTrainer<half>::trainingGPU(Parameter<half> *parameter, half scale) {
	auto value = parameter->value;
	auto gradient = parameter->gradient;

	int N = (int)value.size();
	int blockSize = 1024;
	int grideSize = (N + blockSize - 1) / blockSize;

	SGDTrainerKernel<half> << <grideSize, blockSize >> > (gradient.data(), scale, this->learningRate, value.data(), N);
}
#endif


}