#include "hip/hip_runtime.h"
#include "Exception.h"
#include "GPUException.h"
#include "GPUMathUtils.h"
#include "GPUDevice.h"
#include "SumElements.h"

namespace Deep8 {

#ifdef HAVE_CUDA

template <int blockSize, typename real>
__global__ void SumElementsForwardKernel(const real *x, real *y, const int batch, const int size) {
    SharedMemory<real> shareMemory;
    real *shared = shareMemory.pointer();

    int threaId = threadIdx.x;
    int blockId = blockIdx.x;

    int i = blockId * size + threaId;
    int j = threaId;

    shared[threaId] = 0;

    while (j < size) {
        shared[threaId] += x[i];

        j += blockSize;
        i += blockSize;
    }

    __syncthreads();

    if (blockSize >= 1024) {
        if (threaId < 512) {
            shared[threaId] += shared[threaId + 512];
        }

        __syncthreads();
    }

    if (blockSize >= 512) {
        if (threaId < 256) {
            shared[threaId] += shared[threaId + 256];
        }

        __syncthreads();
    }

    if (blockSize >= 256) {
        if (threaId < 128) {
            shared[threaId] += shared[threaId + 128];
        }

        __syncthreads();
    }

    if (blockSize >= 128) {
        if (threaId < 64) {
            shared[threaId] += shared[threaId + 64];
        }

        __syncthreads();
    }

    if (threaId < 32) {
        warpSumReduce<blockSize, real>(shared, threaId);
    }

    if (0 == threaId) {
        y[blockId] = shared[threaId];
    }
}

template <typename real>
__global__ void SumElementsBackwardKernel(real *xGrad, const real *yGrad, const int size, const int N) {
    int start  = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = start; i < N; i += stride) {
        xGrad[i] += yGrad[i / size];
    }
}

template <typename T>
void SumElements<T>::forwardGPUImpl(Device *device, const T *x, T *y, const int batch, const int size) {
    int blockSize = 1024;

    if (size < blockSize) {
        blockSize = prevPowerOf2(size);
    }

    int sharedSize = sizeof(T) * blockSize;

    if (1024 == blockSize) {
        SumElementsForwardKernel<1024, T> << <batch, blockSize, sharedSize >> > (x, y, batch, size);
    } else if (512 == blockSize) {
        SumElementsForwardKernel<512, T> << <batch, blockSize, sharedSize >> > (x, y, batch, size);
    } else if (256 == blockSize) {
        SumElementsForwardKernel<256, T> << <batch, blockSize, sharedSize >> > (x, y, batch, size);
    } else if (128 == blockSize) {
        SumElementsForwardKernel<128, T> << <batch, blockSize, sharedSize >> > (x, y, batch, size);
    } else if (64 == blockSize) {
        SumElementsForwardKernel<64, T> << <batch, blockSize, sharedSize >> > (x, y, batch, size);
    } else if (32 == blockSize) {
        SumElementsForwardKernel<32, T> << <batch, blockSize, sharedSize >> > (x, y, batch, size);
    } else if (16 == blockSize) {
        SumElementsForwardKernel<16, T> << <batch, blockSize, sharedSize >> > (x, y, batch, size);
    } else if (8 == blockSize) {
        SumElementsForwardKernel<8, T> << <batch, blockSize, sharedSize >> > (x, y, batch, size);
    } else if (4 == blockSize) {
        SumElementsForwardKernel<4, T> << <batch, blockSize, sharedSize >> > (x, y, batch, size);
    } else if (2 == blockSize) {
        SumElementsForwardKernel<2, T> << <batch, blockSize, sharedSize >> > (x, y, batch, size);
    } else if (1 == blockSize) {
        SumElementsForwardKernel<1, T> << <batch, blockSize, sharedSize >> > (x, y, batch, size);
    } else {
        DEEP8_RUNTIME_ERROR("the block size is error");
    }
}

template <typename T>
void SumElements<T>::forwardGPU(const std::vector<const Tensor<T>*> &inputs, Tensor<T> *output) {
    int batch = (int)inputs[0]->shape.batch();
    int size  = (int)inputs[0]->shape.size() / batch;

    forwardGPUImpl(output->device(), inputs[0]->data(), output->data(), batch, size);
}

template <typename T>
void SumElements<T>::backwardGPU(const std::vector<const Tensor<T>*> &inputs,
                                 const Tensor<T> *output,
                                 const Tensor<T> *outputGradient,
                                 size_t index,
                                 Tensor<T> *iGradient) {
    DEEP8_ARGUMENT_CHECK(0 == index, "the index of SumElements backwardCPU is error");

    auto shape = iGradient->shape;
    int N      = (int)shape.size();
    int batch  = (int)shape.batch();
    int size   = N / batch;

    auto dx = iGradient->data();
    auto dy = outputGradient->data();

    int minGrideSize;
    int blockSize;
    int grideSize;

    CUDA_CHECK(hipOccupancyMaxPotentialBlockSize(&minGrideSize, &blockSize, SumElementsBackwardKernel<T>, 0, N));

    grideSize = (N + blockSize - 1) / blockSize;

    SumElementsBackwardKernel<T> << <grideSize, blockSize >> > (dx, dy, size, N);
}

#ifdef HAVE_HALF
template <>
void SumElements<half>::backwardGPU(const std::vector<const Tensor<half>*> &inputs,
                                     const Tensor<half> *output,
                                     const Tensor<half> *outputGradient,
                                     size_t index,
                                     Tensor<half> *iGradient) {
    DEEP8_ARGUMENT_CHECK(0 == index, "the index of SumElements backwardCPU is error");

    auto shape = iGradient->shape;
    int N      = (int)shape.size();
    int batch  = (int)shape.batch();
    int size   = N / batch;

    auto dx = iGradient->data();
    auto dy = outputGradient->data();

    int blockSize = 1024;
    int grideSize = (N + blockSize - 1) / blockSize;

    SumElementsBackwardKernel<T> << <grideSize, blockSize >> > (dx, dy, size, N);
}
#endif

DEEP8_DECLARATION_GPU_FUNC(SumElements);

#endif
}