#include "hip/hip_runtime.h"
#include "Exception.h"
#include "GPUException.h"
#include "GPUMathUtils.h"
#include "GPUDevice.h"
#include "Multiply.h"

namespace Deep8 {

#ifdef HAVE_CUDA

template <typename real>
__global__ void MultiplyForwardKernel(const real *x, const int *xshape, const int *xdims,
                                      const real *y, const int *yshape, const int *ydims,
                                      real *z, const int *zshape, const int *zdims, const int N) {
    int start = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    int zIndex[MAX_TENSOR_DIMS];

    for (int i = start; i < N; i += stride) {
        for (int k = 0, index = i; k < MAX_TENSOR_DIMS; ++k) {
            zIndex[k] = index / zdims[k];
            index %= zdims[k];
        }

        int xIndex = 0;
        int yIndex = 0;

        for (int k = 0; k < MAX_TENSOR_DIMS; ++k) {
            if (xshape[k] == zshape[k]) {
                xIndex += zIndex[k] * xdims[k];
            }

            if (yshape[k] == zshape[k]) {
                yIndex += zIndex[k] * ydims[k];
            }
        }

        z[i] = x[xIndex] * y[yIndex];
    }
}

template <typename real>
__global__ void MultiplyBackwardKernel(real *inGrad,  const int *inShape,  const int *inDims,
                                       const real *otherValue, const int *otherShape, const int *otherDims,
                                       const real *outGrad, const int *outShape, const int *outDims, const int N) {
    int start  = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    int inIndex[MAX_TENSOR_DIMS];
    int otherIndex[MAX_TENSOR_DIMS];
    int outIndex[MAX_TENSOR_DIMS];

    for (int inI = start; inI < N; inI += stride) {
        for (int k = 0, index = inI; k < MAX_TENSOR_DIMS; ++k) {
            inIndex[k]  = index / inDims[k];
            outIndex[k] = inIndex[k];

            index %= inDims[k];
        }

        int j = MAX_TENSOR_DIMS - 1;

        while (j >= 0) {
            if (j == MAX_TENSOR_DIMS - 1) {
                for (int l = 0; l < MAX_TENSOR_DIMS; ++l) {
                    if (otherShape[l] == outShape[l]) {
                        otherIndex[l] = outIndex[l];
                    } else {
                        otherIndex[l] = 0;
                    }
                }

                int otherI = 0;
                int outI = 0;

                for (int l = 0; l < MAX_TENSOR_DIMS; ++l) {
                    otherI += otherIndex[l] * otherDims[l];
                    outI += outIndex[l] * outDims[l];
                }

                inGrad[inI] += otherValue[otherI] * outGrad[outI];
            }

            if (inShape[j] == outShape[j]) {
                j--;
            } else {
                outIndex[j]++;

                if (outIndex[j] >= outShape[j]) {
                    j--;
                } else {
                    for (int l = j + 1; l < MAX_TENSOR_DIMS; ++l) {
                        outIndex[l] = inIndex[l];
                    }

                    j = MAX_TENSOR_DIMS - 1;
                }
            }
        }
    }
}

template <typename T>
void Multiply<T>::forwardGPUImpl(const T *x, const int *xshape, const int *xdims,
                               const T *y, const int *yshape, const int *ydims,
                                     T *z, const int *zshape, const int *zdims, const int N) {
    int minGrideSize;
    int blockSize;
    int grideSize;

    CUDA_CHECK(hipOccupancyMaxPotentialBlockSize(&minGrideSize, &blockSize, MultiplyForwardKernel<T>, 0, N));

    grideSize = (N + blockSize - 1) / blockSize;

    MultiplyForwardKernel<T> << <grideSize, blockSize >> > (x, xshape, xdims, y, yshape, ydims, z, zshape, zdims, N);
}

#ifdef HAVE_HALF
template <>
void Multiply<half>::forwardGPUImpl(const half *x, const int *xshape, const int *xdims,
                                    const half *y, const int *yshape, const int *ydims,
                                          half *z, const int *zshape, const int *zdims, const int N) {
    int blockSize = 1024;
    int grideSize = (N + blockSize - 1) / blockSize;

    MultiplyForwardKernel<half> << <grideSize, blockSize >> > (x, xshape, xdims, y, yshape, ydims, z, zshape, zdims, N);
}
#endif

template <typename T>
void Multiply<T>::forwardGPU(const std::vector<const Tensor<T>*> &inputs, Tensor<T> *output) {
    auto device = static_cast<GPUDevice*>(output->device());

    auto x = inputs[0];
    auto y = inputs[1];
    auto z = output;

    int xshape[MAX_TENSOR_DIMS];
    int yshape[MAX_TENSOR_DIMS];
    int zshape[MAX_TENSOR_DIMS];

    int xdims[MAX_TENSOR_DIMS];
    int ydims[MAX_TENSOR_DIMS];
    int zdims[MAX_TENSOR_DIMS];

    enlongateShapeToMaxDim(x->shape, xshape);
    enlongateShapeToMaxDim(y->shape, yshape);
    enlongateShapeToMaxDim(z->shape, zshape);

    xdims[MAX_TENSOR_DIMS - 1] = 1;
    ydims[MAX_TENSOR_DIMS - 1] = 1;
    zdims[MAX_TENSOR_DIMS - 1] = 1;

    for (int i = MAX_TENSOR_DIMS - 2; i >= 0; --i) {
        xdims[i] = xdims[i + 1] * xshape[i + 1];
        ydims[i] = ydims[i + 1] * yshape[i + 1];
        zdims[i] = zdims[i + 1] * zshape[i + 1];
    }

    auto cudaPtr = (int*)device->malloc(sizeof(int) * MAX_TENSOR_DIMS * 6);

    int *xshapePtr = cudaPtr;
    int *yshapePtr = xshapePtr + MAX_TENSOR_DIMS;
    int *zshapePtr = yshapePtr + MAX_TENSOR_DIMS;
    int *xdimsPtr = zshapePtr + MAX_TENSOR_DIMS;
    int *ydimsPtr = xdimsPtr + MAX_TENSOR_DIMS;
    int *zdimsPtr = ydimsPtr + MAX_TENSOR_DIMS;

    device->copyFromCPUToGPU(xshape, xshapePtr, sizeof(int) * MAX_TENSOR_DIMS);
    device->copyFromCPUToGPU(yshape, yshapePtr, sizeof(int) * MAX_TENSOR_DIMS);
    device->copyFromCPUToGPU(zshape, zshapePtr, sizeof(int) * MAX_TENSOR_DIMS);
    device->copyFromCPUToGPU(xdims, xdimsPtr, sizeof(int) * MAX_TENSOR_DIMS);
    device->copyFromCPUToGPU(ydims, ydimsPtr, sizeof(int) * MAX_TENSOR_DIMS);
    device->copyFromCPUToGPU(zdims, zdimsPtr, sizeof(int) * MAX_TENSOR_DIMS);

    forwardGPUImpl(x->data(), xshapePtr, xdimsPtr, y->data(), yshapePtr, ydimsPtr, z->data(), zshapePtr, zdimsPtr, static_cast<int>(z->shape.size()));

    device->free(cudaPtr);
}

template <typename T>
void Multiply<T>::backwardGPUImpl(T *inGrad,     const int *inShape,    const int *inDims,
                            const T *otherValue, const int *otherShape, const int *otherDims,
                            const T *outGrad,    const int *outShape,   const int *outDims, const int N) {
    int minGrideSize;
    int blockSize;
    int grideSize;

    CUDA_CHECK(hipOccupancyMaxPotentialBlockSize(&minGrideSize, &blockSize, MultiplyBackwardKernel<T>, 0, N));

    grideSize = (N + blockSize - 1) / blockSize;

    MultiplyBackwardKernel<T> << <grideSize, blockSize >> > (inGrad, inShape, inDims, otherValue, otherShape, otherDims, outGrad, outShape, outDims, N);
}

#ifdef HAVE_HALF
template <>
void Multiply<half>::backwardGPUImpl(half *inGrad, const int *inShape, const int *inDims,
                                const half *otherValue, const int *otherShape, const int *otherDims,
                                const half *outGrad, const int *outShape, const int *outDims, const int N) {
    int blockSize = 1024;
    int grideSize = (N + blockSize - 1) / blockSize;

    MultiplyBackwardKernel<half> << <grideSize, blockSize >> > (inGrad, inShape, inDims, otherValue, otherShape, otherDims, outGrad, outShape, outDims, N);
}
#endif

template <typename T>
void Multiply<T>::backwardGPU(const std::vector<const Tensor<T>*> &inputs,
                             const Tensor<T> *output,
                             const Tensor<T> *outputGradient,
                             size_t index,
                             Tensor<T> *iGradient) {
    DEEP8_ARGUMENT_CHECK(0 == index || 1 == index, "the index is error");

    auto device = static_cast<GPUDevice*>(iGradient->device());

    int inShape[MAX_TENSOR_DIMS];
    int otherShape[MAX_TENSOR_DIMS];
    int outShape[MAX_TENSOR_DIMS];

    int inDims[MAX_TENSOR_DIMS];
    int otherDims[MAX_TENSOR_DIMS];
    int outDims[MAX_TENSOR_DIMS];

    enlongateShapeToMaxDim(iGradient->shape, inShape);
    enlongateShapeToMaxDim(outputGradient->shape, outShape);

    if (0 == index) {
        enlongateShapeToMaxDim(inputs[1]->shape, otherShape);
    } else {
        enlongateShapeToMaxDim(inputs[0]->shape, otherShape);
    }

    inDims[MAX_TENSOR_DIMS - 1]    = 1;
    otherDims[MAX_TENSOR_DIMS - 1] = 1;
    outDims[MAX_TENSOR_DIMS - 1] = 1;

    for (int i = MAX_TENSOR_DIMS - 2; i >= 0; --i) {
        inDims[i]    = inDims[i + 1] * inShape[i + 1];
        otherDims[i] = otherDims[i + 1] * otherShape[i + 1];
        outDims[i]   = outDims[i + 1] * outShape[i + 1];
    }

    auto cudaPtr = (int*)device->malloc(sizeof(int) * MAX_TENSOR_DIMS * 6);

    int *inShapePtr    = cudaPtr;
    int *otherShapePtr = inShapePtr + MAX_TENSOR_DIMS;
    int *outShapePtr   = otherShapePtr + MAX_TENSOR_DIMS;

    int *inDimsPtr    = outShapePtr + MAX_TENSOR_DIMS;
    int *otherDimsPtr = inDimsPtr + MAX_TENSOR_DIMS;
    int *outDimsPtr   = otherDimsPtr + MAX_TENSOR_DIMS;

    device->copyFromCPUToGPU(inShape, inShapePtr, sizeof(int) * MAX_TENSOR_DIMS);
    device->copyFromCPUToGPU(otherShape, otherShapePtr, sizeof(int) * MAX_TENSOR_DIMS);
    device->copyFromCPUToGPU(outShape, outShapePtr, sizeof(int) * MAX_TENSOR_DIMS);

    device->copyFromCPUToGPU(inDims, inDimsPtr, sizeof(int) * MAX_TENSOR_DIMS);
    device->copyFromCPUToGPU(otherDims, otherDimsPtr, sizeof(int) * MAX_TENSOR_DIMS);
    device->copyFromCPUToGPU(outDims, outDimsPtr, sizeof(int) * MAX_TENSOR_DIMS);

    if (0 == index) {
        backwardGPUImpl(iGradient->data(), inShapePtr, inDimsPtr,
                        inputs[1]->data(), otherShapePtr, otherDimsPtr,
                        outputGradient->data(), outShapePtr, outDimsPtr,
                        iGradient->size());
    } else {
        backwardGPUImpl(iGradient->data(), inShapePtr, inDimsPtr,
                        inputs[0]->data(), otherShapePtr, otherDimsPtr,
                        outputGradient->data(), outShapePtr, outDimsPtr,
                        iGradient->size());
    }

    device->free(cudaPtr);
}

DEEP8_DECLARATION_GPU_FUNC(Multiply);

#endif

}