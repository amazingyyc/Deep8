#include "hip/hip_runtime.h"
#include "Square.h"

namespace Deep8 {

#ifdef HAVE_CUDA

template <typename real>
__global__ void SquareForwardKernel(const real *X, real *Y, const int N) {
    int start  = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = start; i < N; i += stride) {
        Y[i] = X[i] * X[i];
    }
}

template <typename real>
__global__ void SquareBackwardKernel(real *xGrad, const real *X, const real *yGrad, const int N) {
    int start = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = start; i < N; i += stride) {
        xGrad[i] += real(2.0) * yGrad[i] * X[i];
    }
}

template <typename T>
void Square<T>::forwardGPU(const std::vector<const Tensor<T>*> &inputs, Tensor<T> *output) {
    auto x = inputs[0]->data();
    auto y = output->data();
    auto N = (int)output->size();

    int minGrideSize;
    int blockSize;
    int grideSize;

    CUDA_CHECK(hipOccupancyMaxPotentialBlockSize(&minGrideSize, &blockSize, SquareForwardKernel<T>, 0, N));

    grideSize = (N + blockSize - 1) / blockSize;

    SquareForwardKernel<T> << <grideSize, blockSize >> > (x, y, N);
}

#ifdef HAVE_HALF
template <>
void Square<half>::forwardGPU(const std::vector<const Tensor<half>*> &inputs, Tensor<half> *output) {
    auto x = inputs[0]->data();
    auto y = output->data();
    auto N = (int)output->size();

    int blockSize = 1024;
    int grideSize = (N + blockSize - 1) / blockSize;

    SquareForwardKernel<half> << <grideSize, blockSize >> > (x, y, N);
}
#endif

template <typename T>
void Square<T>::backwardGPU(const std::vector<const Tensor<T>*> &inputs,
                             const Tensor<T> *output,
                             const Tensor<T> *outputGradient,
                             size_t index,
                             Tensor<T> *iGradient) {
    DEEP8_ARGUMENT_CHECK(0 == index, "the index is error");

    auto dx = iGradient->data();
    auto x  = inputs[0]->data();
    auto dy = outputGradient->data();
    auto N  = (int) iGradient->size();

    int minGrideSize;
    int blockSize;
    int grideSize;

    CUDA_CHECK(hipOccupancyMaxPotentialBlockSize(&minGrideSize, &blockSize, SquareBackwardKernel<T>, 0, N));

    grideSize = (N + blockSize - 1) / blockSize;

    SquareBackwardKernel<T> << <grideSize, blockSize >> > (dx, x, dy, N);
}

#ifdef HAVE_HALF
template <>
void Square<half>::backwardGPU(const std::vector<const Tensor<half>*> &inputs,
                             const Tensor<half> *output,
                             const Tensor<half> *outputGradient,
                             size_t index,
                             Tensor<half> *iGradient) {
    DEEP8_ARGUMENT_CHECK(0 == index, "the index is error");

    auto dx = iGradient->data();
    auto x  = inputs[0]->data();
    auto dy = outputGradient->data();
    auto N  = (int) iGradient->size();

    int blockSize = 1024;
    int grideSize = (N + blockSize - 1) / blockSize;

    SquareBackwardKernel<half> << <grideSize, blockSize >> > (dx, x, dy, N);
}
#endif

DEEP8_DECLARATION_GPU_FUNC(Square);

#endif

}