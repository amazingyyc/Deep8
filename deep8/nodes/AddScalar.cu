#include "hip/hip_runtime.h"
#include "Exception.h"
#include "GPUException.h"
#include "GPUMathUtils.h"
#include "GPUDevice.h"
#include "AddScalar.h"

namespace Deep8 {

#ifdef HAVE_CUDA

template <typename real>
__global__ void AddScalarForwardKernel(const real *x, const real scalar, real *y, const int N) {
    int start  = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = start; i < N; i += stride) {
        y[i] = x[i] + scalar;
    }
}

template <typename real>
__global__ void AddScalarBackwardKernel(real *xGrad, const real *yGrad, const int N) {
    int start  = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = start; i < N; i += stride) {
        xGrad[i] += yGrad[i];
    }
}

template <typename T>
void AddScalar<T>::forwardGPU(const std::vector<const Tensor<T>*> &inputs, Tensor<T> *output) {
    auto x = inputs[0]->data();
    auto y = output->data();
    auto N = static_cast<int>(output->size());

    int minGrideSize;
    int blockSize;
    int grideSize;

    CUDA_CHECK(hipOccupancyMaxPotentialBlockSize(&minGrideSize, &blockSize, AddScalarForwardKernel<T>, 0, N));

    grideSize = (N + blockSize - 1) / blockSize;

    AddScalarForwardKernel<T> << <grideSize, blockSize >> > (X, scalar, Y, N);
}

#ifdef HAVE_HALF
template <>
void AddScalar<half>::forwardGPU(const std::vector<const Tensor<half>*> &inputs, Tensor<half> *output) {
    auto x = inputs[0]->data();
    auto y = output->data();
    auto N = static_cast<int>(output->size());

    int blockSize = 1024;
    int grideSize = (N + blockSize - 1) / blockSize;

    AddScalarForwardKernel<half> << <grideSize, blockSize >> > (X, scalar, Y, N);
}
#endif

template <>
void AddScalar<float>::backwardGPU(const std::vector<const Tensor<float>*> &inputs, const Tensor<float> *output, const Tensor<float> *outputGradient, size_t index, Tensor<float> *iGradient) {
    DEEP8_ARGUMENT_CHECK(0 == index, "the index is error");

    auto device = static_cast<GPUDevice*>(iGradient->device());

    float alpha = 1;

    CUBLAS_CHECK(hipblasSaxpy(device->cublasHandle, (int)iGradient->size(), &alpha, outputGradient->data(), 1, iGradient->data(), 1));
}

template <>
void AddScalar<double>::backwardGPU(const std::vector<const Tensor<double>*> &inputs, const Tensor<double> *output, const Tensor<double> *outputGradient, size_t index, Tensor<double> *iGradient) {
    DEEP8_ARGUMENT_CHECK(0 == index, "the index is error");

    auto device = static_cast<GPUDevice*>(iGradient->device());

    double alpha = 1;

    CUBLAS_CHECK(hipblasDaxpy(device->cublasHandle, (int)iGradient->size(), &alpha, outputGradient->data(), 1, iGradient->data(), 1));
}

#ifdef HAVE_HALF
template <>
void AddScalar<half>::backwardGPU(const std::vector<const Tensor<half>*> &inputs, const Tensor<half> *output, const Tensor<half> *outputGradient, size_t index, Tensor<half> *iGradient) {
    DEEP8_ARGUMENT_CHECK(0 == index, "the index is error");

    auto device = (GPUDevice*)(iGradient->device());
    auto N      = (int)iGradient->size();

    int blockSize = 1024;
    int grideSize = (N + blockSize - 1) / blockSize;

    AddScalarBackwardKernel<half> << <grideSize, blockSize >> > (iGradient->data(), outputGradient->data(), N);
}
#endif

DEEP8_DECLARATION_GPU_FUNC(AddScalar);

#endif

}