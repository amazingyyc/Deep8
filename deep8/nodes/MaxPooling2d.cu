#include "hip/hip_runtime.h"
#include "MaxPooling2d.h"

namespace Deep8 {

#ifdef HAVE_CUDA

#ifdef HAVE_CUDNN
template <>
void MaxPooling2d<float>::forwardGPUCUDNNImpl(Device *d, const float *x, const Shape &xShape, float *y, const Shape &yShape,
                         int windowsHeight,   int windowsWidth,
                         int verticalPadding, int horizontalPadding,
                         int verticalStride,  int horizontalStride) {
    auto device = (GPUDevice*)d;

    float alpha = 1;
    float beta  = 0;

    hipdnnPoolingDescriptor_t poolingDesc;
    CUDNN_CHECK(hipdnnCreatePoolingDescriptor(&poolingDesc));
    CUDNN_CHECK(hipdnnSetPooling2dDescriptor(poolingDesc, HIPDNN_POOLING_MAX, HIPDNN_PROPAGATE_NAN,
                                            windowsHeight, windowsWidth, verticalPadding, horizontalPadding, verticalStride, horizontalStride));

    hipdnnTensorDescriptor_t xDesc;
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&xDesc));
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(xDesc, HIPDNN_TENSOR_NHWC, HIPDNN_DATA_FLOAT, (int)xShape.dim(0), (int)xShape.dim(3), (int)xShape.dim(1), (int)xShape.dim(2)));

    hipdnnTensorDescriptor_t yDesc;
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&yDesc));
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(yDesc, HIPDNN_TENSOR_NHWC, HIPDNN_DATA_FLOAT, (int)yShape.dim(0), (int)yShape.dim(3), (int)yShape.dim(1), (int)yShape.dim(2)));

    CUDNN_CHECK(hipdnnPoolingForward(device->cudnnHandle, poolingDesc, &alpha, xDesc, x, &beta, yDesc, y));

    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(yDesc));
    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(xDesc));
    CUDNN_CHECK(hipdnnDestroyPoolingDescriptor(poolingDesc));
}

template <>
void MaxPooling2d<double>::forwardGPUCUDNNImpl(Device *d, const double *x, const Shape &xShape, double *y, const Shape &yShape,
                         int windowsHeight,   int windowsWidth,
                         int verticalPadding, int horizontalPadding,
                         int verticalStride,  int horizontalStride) {
    auto device = (GPUDevice*)d;

    double alpha = 1;
    double beta  = 0;

    hipdnnPoolingDescriptor_t poolingDesc;
    CUDNN_CHECK(hipdnnCreatePoolingDescriptor(&poolingDesc));
    CUDNN_CHECK(hipdnnSetPooling2dDescriptor(poolingDesc, HIPDNN_POOLING_MAX, HIPDNN_PROPAGATE_NAN,
                                            windowsHeight, windowsWidth, verticalPadding, horizontalPadding, verticalStride, horizontalStride));

    hipdnnTensorDescriptor_t xDesc;
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&xDesc));
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(xDesc, HIPDNN_TENSOR_NHWC, HIPDNN_DATA_DOUBLE, (int)xShape.dim(0), (int)xShape.dim(3), (int)xShape.dim(1), (int)xShape.dim(2)));

    hipdnnTensorDescriptor_t yDesc;
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&yDesc));
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(yDesc, HIPDNN_TENSOR_NHWC, HIPDNN_DATA_DOUBLE, (int)yShape.dim(0), (int)yShape.dim(3), (int)yShape.dim(1), (int)yShape.dim(2)));

    CUDNN_CHECK(hipdnnPoolingForward(device->cudnnHandle, poolingDesc, &alpha, xDesc, x, &beta, yDesc, y));

    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(yDesc));
    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(xDesc));
    CUDNN_CHECK(hipdnnDestroyPoolingDescriptor(poolingDesc));
}

#ifdef HAVE_HALF
template <>
void MaxPooling2d<half>::forwardGPUCUDNNImpl(Device *d, const half *x, const Shape &xShape, half *y, const Shape &yShape,
                        int windowsHeight, int windowsWidth,
                        int verticalPadding, int horizontalPadding,
                        int verticalStride, int horizontalStride) {
    auto device = (GPUDevice*)d;

    half alpha = 1;
    half beta = 0;

    hipdnnPoolingDescriptor_t poolingDesc;
    CUDNN_CHECK(hipdnnCreatePoolingDescriptor(&poolingDesc));
    CUDNN_CHECK(hipdnnSetPooling2dDescriptor(poolingDesc, HIPDNN_POOLING_MAX, HIPDNN_PROPAGATE_NAN,
        windowsHeight, windowsWidth, verticalPadding, horizontalPadding, verticalStride, horizontalStride));

    hipdnnTensorDescriptor_t xDesc;
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&xDesc));
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(xDesc, HIPDNN_TENSOR_NHWC, HIPDNN_DATA_HALF, (int)xShape.dim(0), (int)xShape.dim(3), (int)xShape.dim(1), (int)xShape.dim(2)));

    hipdnnTensorDescriptor_t yDesc;
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&yDesc));
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(yDesc, HIPDNN_TENSOR_NHWC, HIPDNN_DATA_HALF, (int)yShape.dim(0), (int)yShape.dim(3), (int)yShape.dim(1), (int)yShape.dim(2)));

    CUDNN_CHECK(hipdnnPoolingForward(device->cudnnHandle, poolingDesc, &alpha, xDesc, x, &beta, yDesc, y));

    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(yDesc));
    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(xDesc));
    CUDNN_CHECK(hipdnnDestroyPoolingDescriptor(poolingDesc));
}
#endif
#endif

template <typename T>
void MaxPooling2d<T>::forwardGPU(const std::vector<const Tensor<T>*> &inputs, Tensor<T> *output)  {
#ifdef HAVE_CUDNN

    auto inputH = static_cast<int>(inputs[0]->shape.dim(1));
    auto inputW = static_cast<int>(inputs[0]->shape.dim(2));

    auto outputH = static_cast<int>(output->shape.dim(1));
    auto outputW = static_cast<int>(output->shape.dim(2));

    int padY = std::max<int>(0, (outputH - 1) * static_cast<int>(strideY) + static_cast<int>(filterHeight) - inputH);
    int padX = std::max<int>(0, (outputW - 1) * static_cast<int>(strideX) + static_cast<int>(filterWidth) - inputW);

    int padTop  = (padY / 2);
    int padLeft = (padX / 2);

    forwardGPUCUDNNImpl(output->device(), inputs[0]->data(), inputs[0]->shape, output->data(), output->shape,
        (int)filterHeight, (int)filterWidth, padTop, padLeft, (int)strideY, (int)strideX);
#else
    DEEP8_RUNTIME_ERROR("the MaxPooling2d needs CUDNN");
#endif
}


#ifdef HAVE_CUDNN

template <>
void MaxPooling2d<float>::backwardGPUCUDNNImpl(Device *d, const float *x, float *dx, const Shape &xShape, const float *y, const float *dy, const Shape &yShape,
                          int windowsHeight, int windowsWidth,
                          int verticalPadding, int horizontalPadding,
                          int verticalStride, int horizontalStride) {
    auto device = (GPUDevice*)d;

    float alpha = 1;
    float beta = 1;

    hipdnnPoolingDescriptor_t poolingDesc;
    CUDNN_CHECK(hipdnnCreatePoolingDescriptor(&poolingDesc));
    CUDNN_CHECK(hipdnnSetPooling2dDescriptor(poolingDesc, HIPDNN_POOLING_MAX, HIPDNN_PROPAGATE_NAN,
                                            windowsHeight, windowsWidth, verticalPadding, horizontalPadding, verticalStride, horizontalStride));

    hipdnnTensorDescriptor_t xDesc;
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&xDesc));
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(xDesc, HIPDNN_TENSOR_NHWC, HIPDNN_DATA_FLOAT, (int)xShape.dim(0), (int)xShape.dim(3), (int)xShape.dim(1), (int)xShape.dim(2)));

    hipdnnTensorDescriptor_t dxDesc;
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&dxDesc));
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(dxDesc, HIPDNN_TENSOR_NHWC, HIPDNN_DATA_FLOAT, (int)xShape.dim(0), (int)xShape.dim(3), (int)xShape.dim(1), (int)xShape.dim(2)));

    hipdnnTensorDescriptor_t yDesc;
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&yDesc));
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(yDesc, HIPDNN_TENSOR_NHWC, HIPDNN_DATA_FLOAT, (int)yShape.dim(0), (int)yShape.dim(3), (int)yShape.dim(1), (int)yShape.dim(2)));

    hipdnnTensorDescriptor_t dyDesc;
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&dyDesc));
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(dyDesc, HIPDNN_TENSOR_NHWC, HIPDNN_DATA_FLOAT, (int)yShape.dim(0), (int)yShape.dim(3), (int)yShape.dim(1), (int)yShape.dim(2)));

    CUDNN_CHECK(hipdnnPoolingBackward(device->cudnnHandle, poolingDesc, &alpha, yDesc, y, dyDesc, dy, xDesc, x, &beta, dxDesc, dx));

    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(dyDesc));
    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(yDesc));
    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(dxDesc));
    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(xDesc));
    CUDNN_CHECK(hipdnnDestroyPoolingDescriptor(poolingDesc));
}

template <>
void MaxPooling2d<double>::backwardGPUCUDNNImpl(Device *d, const double *x, double *dx, const Shape &xShape, const double *y, const double *dy, const Shape &yShape,
                          int windowsHeight, int windowsWidth,
                          int verticalPadding, int horizontalPadding,
                          int verticalStride, int horizontalStride) {
    auto device = (GPUDevice*)d;

    double alpha = 1;
    double beta = 1;

    hipdnnPoolingDescriptor_t poolingDesc;
    CUDNN_CHECK(hipdnnCreatePoolingDescriptor(&poolingDesc));
    CUDNN_CHECK(hipdnnSetPooling2dDescriptor(poolingDesc, HIPDNN_POOLING_MAX, HIPDNN_PROPAGATE_NAN,
                                            windowsHeight, windowsWidth, verticalPadding, horizontalPadding, verticalStride, horizontalStride));

    hipdnnTensorDescriptor_t xDesc;
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&xDesc));
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(xDesc, HIPDNN_TENSOR_NHWC, HIPDNN_DATA_DOUBLE, (int)xShape.dim(0), (int)xShape.dim(3), (int)xShape.dim(1), (int)xShape.dim(2)));

    hipdnnTensorDescriptor_t dxDesc;
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&dxDesc));
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(dxDesc, HIPDNN_TENSOR_NHWC, HIPDNN_DATA_DOUBLE, (int)xShape.dim(0), (int)xShape.dim(3), (int)xShape.dim(1), (int)xShape.dim(2)));

    hipdnnTensorDescriptor_t yDesc;
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&yDesc));
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(yDesc, HIPDNN_TENSOR_NHWC, HIPDNN_DATA_DOUBLE, (int)yShape.dim(0), (int)yShape.dim(3), (int)yShape.dim(1), (int)yShape.dim(2)));

    hipdnnTensorDescriptor_t dyDesc;
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&dyDesc));
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(dyDesc, HIPDNN_TENSOR_NHWC, HIPDNN_DATA_DOUBLE, (int)yShape.dim(0), (int)yShape.dim(3), (int)yShape.dim(1), (int)yShape.dim(2)));

    CUDNN_CHECK(hipdnnPoolingBackward(device->cudnnHandle, poolingDesc, &alpha, yDesc, y, dyDesc, dy, xDesc, x, &beta, dxDesc, dx));

    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(dyDesc));
    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(yDesc));
    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(dxDesc));
    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(xDesc));
    CUDNN_CHECK(hipdnnDestroyPoolingDescriptor(poolingDesc));
}

#ifdef HAVE_HALF
template <>
void MaxPooling2d<half>::backwardGPUCUDNNImpl(Device *d, const half *x, half *dx, const Shape &xShape, const half *y, const half *dy, const Shape &yShape,
                                    int windowsHeight, int windowsWidth,
                                    int verticalPadding, int horizontalPadding,
                                    int verticalStride, int horizontalStride) {
    auto device = (GPUDevice*)d;

    half alpha = 1;
    half beta = 1;

    hipdnnPoolingDescriptor_t poolingDesc;
    CUDNN_CHECK(hipdnnCreatePoolingDescriptor(&poolingDesc));
    CUDNN_CHECK(hipdnnSetPooling2dDescriptor(poolingDesc, HIPDNN_POOLING_MAX, HIPDNN_PROPAGATE_NAN,
        windowsHeight, windowsWidth, verticalPadding, horizontalPadding, verticalStride, horizontalStride));

    hipdnnTensorDescriptor_t xDesc;
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&xDesc));
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(xDesc, HIPDNN_TENSOR_NHWC, HIPDNN_DATA_HALF, (int)xShape.dim(0), (int)xShape.dim(3), (int)xShape.dim(1), (int)xShape.dim(2)));

    hipdnnTensorDescriptor_t dxDesc;
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&dxDesc));
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(dxDesc, HIPDNN_TENSOR_NHWC, HIPDNN_DATA_HALF, (int)xShape.dim(0), (int)xShape.dim(3), (int)xShape.dim(1), (int)xShape.dim(2)));

    hipdnnTensorDescriptor_t yDesc;
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&yDesc));
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(yDesc, HIPDNN_TENSOR_NHWC, HIPDNN_DATA_HALF, (int)yShape.dim(0), (int)yShape.dim(3), (int)yShape.dim(1), (int)yShape.dim(2)));

    hipdnnTensorDescriptor_t dyDesc;
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&dyDesc));
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(dyDesc, HIPDNN_TENSOR_NHWC, HIPDNN_DATA_HALF, (int)yShape.dim(0), (int)yShape.dim(3), (int)yShape.dim(1), (int)yShape.dim(2)));

    CUDNN_CHECK(hipdnnPoolingBackward(device->cudnnHandle, poolingDesc, &alpha, yDesc, y, dyDesc, dy, xDesc, x, &beta, dxDesc, dx));

    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(dyDesc));
    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(yDesc));
    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(dxDesc));
    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(xDesc));
    CUDNN_CHECK(hipdnnDestroyPoolingDescriptor(poolingDesc));
}
#endif
#endif

template <typename T>
void MaxPooling2d<T>::backwardGPU(const std::vector<const Tensor<T>*> &inputs,
                             const Tensor<T> *output,
                             const Tensor<T> *outputGradient,
                             size_t index,
                             Tensor<T> *iGradient) {
#ifdef HAVE_CUDNN

    auto inputH = static_cast<int>(iGradient->shape.dim(1));
    auto inputW = static_cast<int>(iGradient->shape.dim(2));

    auto outputH = static_cast<int>(outputGradient->shape.dim(1));
    auto outputW = static_cast<int>(outputGradient->shape.dim(2));

    int padY = std::max<int>(0, (outputH - 1) * static_cast<int>(strideY) + static_cast<int>(filterHeight) - inputH);
    int padX = std::max<int>(0, (outputW - 1) * static_cast<int>(strideX) + static_cast<int>(filterWidth) - inputW);

    int padTop  = (padY / 2);
    int padLeft = (padX / 2);

    backwardGPUCUDNNImpl(output->device(),
        inputs[0]->data(), iGradient->data(), iGradient->shape,
        output->data(), outputGradient->data(), output->shape,
        (int)filterHeight, (int)filterWidth, padTop, padLeft, (int)strideY, (int)strideX);

#else
    DEEP8_RUNTIME_ERROR("the MaxPooling2d needs CUDNN");
#endif
}

DEEP8_DECLARATION_GPU_FUNC(MaxPooling2d);

#endif

}