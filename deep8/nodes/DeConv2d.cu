#include "hip/hip_runtime.h"
#include "Exception.h"
#include "GPUException.h"
#include "GPUMathUtils.h"
#include "GPUDevice.h"
#include "DeConv2d.h"

namespace Deep8 {

#ifdef HAVE_CUDA

template <typename real>
__global__ void DeConv2dForwardKernel(const real *outputMat, real *output,
    const int batch, const int inputHeight, const int inputWidth, const int inputChannel,
	const int filterHeight, const int filterWidth,
    const int outputHeight, const int outputWidth, const int outputChannel,
    const int forwardStrideY, const int forwardStrideX,
    const int padTop, const int padLeft,
	 const int N) {

    int start  = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = start; i < N; i += stride) {
        int b = i / (outputHeight * outputWidth * outputChannel);

        int outputY = (i % (outputHeight * outputWidth * outputChannel)) / (outputWidth * outputChannel);
        int outputX = (i % (outputWidth * outputChannel)) / outputChannel;
        int outputOffset = i % outputChannel;

        real out = 0;

        for (int y = 0; y < filterHeight; ++y) {
            for (int x = 0; x < filterWidth; ++x) {
                int inputY = outputY + padTop + y;
                int inputX = outputX + padLeft + x;

                if (0 == inputY % forwardStrideY && 0 == inputX % forwardStrideX) {
                    inputY /= forwardStrideY;
                    inputX /= forwardStrideX;

                    if (0 <= inputY && inputY < inputHeight && 0 <= inputX && inputX < inputWidth) {
                        out += outputMat[(b * inputHeight * inputWidth + inputY * inputWidth + inputX) * (outputChannel * filterHeight * filterWidth)
                        + outputOffset * filterHeight * filterWidth + y * filterWidth + x];
                    }
                }
            }
        }

        output[i] = out;
    }
}

template <typename real>
__global__ void DeConv2dBackwardKernel(real *dyMat, const real *dy,
    const int batch, const int inputHeight, const int inputWidth, const int inputChannel,
	const int filterHeight, const int filterWidth,
    const int outputHeight, const int outputWidth, const int outputChannel,
    const int forwardStrideY, const int forwardStrideX,
    const int padTop, const int padLeft, const int N) {

    int start  = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = start; i < N; i += stride) {
        int row = i / (outputChannel * filterHeight * filterWidth);
        int col = i % (outputChannel * filterHeight * filterWidth);

        int b = row / (inputHeight * inputWidth);
        int inputY = (row % (inputHeight * inputWidth)) / inputWidth;
        int inputX = row % inputWidth;

        int outputOffset = col / (filterHeight * filterWidth);
        int filterY = (col % (filterHeight * filterWidth)) / filterWidth;
        int filterX = col % filterWidth;

        int outputY = inputY * forwardStrideY - padTop - filterY;
        int outputX = inputX * forwardStrideX - padLeft - filterX;

        if (0 <= outputY && outputY < outputHeight && 0 <= outputX && outputX < outputWidth) {
            dyMat[i] = dy[((b * outputHeight + outputY) * outputWidth + outputX) * outputChannel + outputOffset];
        } else {
            dyMat[i] = 0;
        }
    }
}

template <>
void DeConv2d<float>::forwardGPUImpl(Device *d, const float *x, const float *filter, float *y,
                    int batch, int inputHeight, int inputWidth, int inputChannel,
                    int outputHeight, int outputWidth, int outputChannel,
                    int filterHeight, int filterWidth, int forwardStrideY, int forwardStrideX,
                    int padTop, int padLeft) {
    auto device = (GPUDevice*)d;

    int m = batch * inputHeight * inputWidth;
    int k = inputChannel;
    int n = outputChannel * filterHeight * filterWidth;

    float alpha = 1;
    float beta  = 0;

    auto yMat = (float*)device->malloc(sizeof(float) * m * n);

    CUBLAS_CHECK(hipblasSgemm(device->cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_N, n, m, k, &alpha, filter, k, x, k, &beta, yMat, n));

    int size = batch * outputHeight * outputWidth * outputChannel;

    int minGrideSize;
    int blockSize;
    int grideSize;

    CUDA_CHECK(hipOccupancyMaxPotentialBlockSize(&minGrideSize, &blockSize, DeConv2dForwardKernel<float>, 0, size));

    grideSize = (size + blockSize - 1) / blockSize;

    DeConv2dForwardKernel<float><<<grideSize, blockSize>>>(yMat, y,
            batch, inputHeight, inputWidth, inputChannel,
            filterHeight, filterWidth,
            outputHeight, outputWidth, outputChannel,
            forwardStrideY, forwardStrideX,
            padTop, padLeft, size);

    device->free(yMat);
}

template <>
void DeConv2d<double>::forwardGPUImpl(Device *d, const double *x, const double *filter, double *y,
                    int batch, int inputHeight, int inputWidth, int inputChannel,
                    int outputHeight, int outputWidth, int outputChannel,
                    int filterHeight, int filterWidth, int forwardStrideY, int forwardStrideX,
                    int padTop, int padLeft) {
    auto device = (GPUDevice*)d;

    int m = batch * inputHeight * inputWidth;
    int k = inputChannel;
    int n = outputChannel * filterHeight * filterWidth;

    double alpha = 1;
    double beta = 0;

    auto yMat = (double*)device->malloc(sizeof(double) * m * n);

    CUBLAS_CHECK(hipblasDgemm(device->cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_N, n, m, k, &alpha, filter, k, x, k, &beta, yMat, n));

    int size = batch * outputHeight * outputWidth * outputChannel;

    int minGrideSize;
    int blockSize;
    int grideSize;

    CUDA_CHECK(hipOccupancyMaxPotentialBlockSize(&minGrideSize, &blockSize, DeConv2dForwardKernel<double>, 0, size));

    grideSize = (size + blockSize - 1) / blockSize;

    DeConv2dForwardKernel<double> << <grideSize, blockSize >> > (yMat, y,
            batch, inputHeight, inputWidth, inputChannel,
            filterHeight, filterWidth,
            outputHeight, outputWidth, outputChannel,
            forwardStrideY, forwardStrideX,
            padTop, padLeft, size);

    device->free(yMat);
}

#ifdef HAVE_HALF

template <>
void DeConv2d<half>::forwardGPUImpl(Device *d, const half *x, const half *filter, half *y,
                            int batch, int inputHeight, int inputWidth, int inputChannel,
                            int outputHeight, int outputWidth, int outputChannel,
                            int filterHeight, int filterWidth, int forwardStrideY, int forwardStrideX,
                            int padTop, int padLeft) {
    auto device = (GPUDevice*)d;
    int m = batch * inputHeight * inputWidth;
    int k = inputChannel;
    int n = outputChannel * filterHeight * filterWidth;

    half alpha = 1;
    half beta = 0;

    auto yMat = (half*)device->malloc(sizeof(half) * m * n);

    CUBLAS_CHECK(hipblasHgemm(device->cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_N, n, m, k, &alpha, filter, k, x, k, &beta, yMat, n));

    int size = batch * outputHeight * outputWidth * outputChannel;

    int blockSize = 1024;
    int grideSize = (size + blockSize - 1) / blockSize;

    DeConv2dForwardKernel<half> << <grideSize, blockSize >> > (yMat, y,
        batch, inputHeight, inputWidth, inputChannel,
        filterHeight, filterWidth,
        outputHeight, outputWidth, outputChannel,
        forwardStrideY, forwardStrideX,
        padTop, padLeft, size);

    device->free(yMat);
}
#endif

template <typename T>
void DeConv2d<T>::forwardGPU(const std::vector<const Tensor<T>*> &inputs, Tensor<T> *output) {
    auto device = static_cast<GPUDevice*>(output->device());

    auto x = inputs[0];
    auto filter = inputs[1];

    auto y = output;

    auto batch        = (int)x->shape.dim(0);
    auto inputHeight  = (int)x->shape.dim(1);
    auto inputWidth   = (int)x->shape.dim(2);
    auto inputChannel = (int)x->shape.dim(3);

    auto outputHeight  = (int)y->shape.dim(1);
    auto outputWidth   = (int)y->shape.dim(2);
    auto outputChannel = (int)y->shape.dim(3);

    auto filterHeight = (int)filter->shape.dim(1);
    auto filterWidth  = (int)filter->shape.dim(2);

    auto padTop  = -(std::max<int>(0, outputHeight + filterHeight - (inputHeight - 1) * (int)(forwardStrideY) - 2) / 2);
    auto padLeft = -(std::max<int>(0, outputWidth  + filterWidth  - (inputWidth  - 1) * (int)(forwardStrideX) - 2) / 2);

    forwardGPUImpl(device, x->data(), filter->data(), y->data(),
                    batch, inputHeight, inputWidth, inputChannel,
                    outputHeight, outputWidth, outputChannel,
                    filterHeight, filterWidth, (int)forwardStrideY, (int)forwardStrideX,
                    padTop, padLeft);
}

template <>
void DeConv2d<float>::backwardGPUInputImpl(Device *d, float *dx, const float *filter, const float *dy,
                          int batch, int inputHeight, int inputWidth, int inputChannel,
                          int outputHeight, int outputWidth, int outputChannel,
                          int filterHeight, int filterWidth, int forwardStrideY, int forwardStrideX,
                          int padTop, int padLeft) {
    auto device = (GPUDevice*)d;
    int size = batch * inputHeight * inputWidth * outputChannel * filterHeight * filterWidth;

    auto dyMat = (float*)device->malloc(sizeof(float) * size);

    int minGrideSize;
    int blockSize;
    int grideSize;

    CUDA_CHECK(hipOccupancyMaxPotentialBlockSize(&minGrideSize, &blockSize, DeConv2dBackwardKernel<float>, 0, size));

    grideSize = (size + blockSize - 1) / blockSize;

    DeConv2dBackwardKernel<float><<<grideSize, blockSize>>>(dyMat, dy,
            batch, inputHeight, inputWidth, inputChannel,
            filterHeight, filterWidth,
            outputHeight, outputWidth, outputChannel,
            forwardStrideY, forwardStrideX,
            padTop, padLeft, size);


    int m = batch * inputHeight * inputWidth;
    int n = outputChannel * filterHeight * filterWidth;
    int k = inputChannel;

    float alpha = 1;
    float beta  = 1;

    CUBLAS_CHECK(hipblasSgemm(device->cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, k,m, n, &alpha, filter, k, dyMat, n, &beta, dx, k));

    device->free(dyMat);
}

template <>
void DeConv2d<double>::backwardGPUInputImpl(Device *d, double *dx, const double *filter, const double *dy,
                          int batch, int inputHeight, int inputWidth, int inputChannel,
                          int outputHeight, int outputWidth, int outputChannel,
                          int filterHeight, int filterWidth, int forwardStrideY, int forwardStrideX,
                          int padTop, int padLeft) {
    auto device = (GPUDevice*)d;
    int size = batch * inputHeight * inputWidth * outputChannel * filterHeight * filterWidth;

    auto dyMat = (double*)device->malloc(sizeof(double) * size);

    int minGrideSize;
    int blockSize;
    int grideSize;

    CUDA_CHECK(hipOccupancyMaxPotentialBlockSize(&minGrideSize, &blockSize, DeConv2dBackwardKernel<double>, 0, size));

    grideSize = (size + blockSize - 1) / blockSize;

    DeConv2dBackwardKernel<double> << <grideSize, blockSize >> > (dyMat, dy,
            batch, inputHeight, inputWidth, inputChannel,
            filterHeight, filterWidth,
            outputHeight, outputWidth, outputChannel,
            forwardStrideY, forwardStrideX,
            padTop, padLeft, size);


    int m = batch * inputHeight * inputWidth;
    int n = outputChannel * filterHeight * filterWidth;
    int k = inputChannel;

    double alpha = 1;
    double beta = 1;

    CUBLAS_CHECK(hipblasDgemm(device->cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, k, m, n, &alpha, filter, k, dyMat, n, &beta, dx, k));

    device->free(dyMat);
}

#ifdef HAVE_HALF
template <>
void DeConv2d<half>::backwardGPUInputImpl(Device *d, half *dx, const half *filter, const half *dy,
		int batch, int inputHeight, int inputWidth, int inputChannel,
		int outputHeight, int outputWidth, int outputChannel,
		int filterHeight, int filterWidth, int forwardStrideY, int forwardStrideX,
		int padTop, int padLeft) {
    auto device = (GPUDevice*)d;
    int size = batch * inputHeight * inputWidth * outputChannel * filterHeight * filterWidth;

    auto dyMat = (half*)device->malloc(sizeof(half) * size);

    int blockSize = 1024;
    int grideSize = (size + blockSize - 1) / blockSize;

    DeConv2dBackwardKernel<half> << <grideSize, blockSize >> > (dyMat, dy,
        batch, inputHeight, inputWidth, inputChannel,
        filterHeight, filterWidth,
        outputHeight, outputWidth, outputChannel,
        forwardStrideY, forwardStrideX,
        padTop, padLeft, size);


    int m = batch * inputHeight * inputWidth;
    int n = outputChannel * filterHeight * filterWidth;
    int k = inputChannel;

    half alpha = 1;
    half beta = 1;

    CUBLAS_CHECK(hipblasHgemm(device->cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, k, m, n, &alpha, filter, k, dyMat, n, &beta, dx, k));

    device->free(dyMat);
}
#endif

template <>
void DeConv2d<float>::backwardGPUFilterImpl(Device *d, const float *x, float *dw, const float *dy,
                           int batch, int inputHeight, int inputWidth, int inputChannel,
                           int outputHeight, int outputWidth, int outputChannel,
                           int filterHeight, int filterWidth, int forwardStrideY, int forwardStrideX,
                           int padTop, int padLeft) {
    auto device = (GPUDevice*)d;
    int size = batch * inputHeight * inputWidth * outputChannel * filterHeight * filterWidth;

    auto dyMat = (float*)device->malloc(sizeof(float) * size);

    int minGrideSize;
    int blockSize;
    int grideSize;

    CUDA_CHECK(hipOccupancyMaxPotentialBlockSize(&minGrideSize, &blockSize, DeConv2dBackwardKernel<float>, 0, size));

    grideSize = (size + blockSize - 1) / blockSize;

    DeConv2dBackwardKernel<float><<<grideSize, blockSize>>>(dyMat, dy,
            batch, inputHeight, inputWidth, inputChannel,
            filterHeight, filterWidth,
            outputHeight, outputWidth, outputChannel,
            forwardStrideY, forwardStrideX,
            padTop, padLeft, size);

    int m = batch * inputHeight * inputWidth;
    int n = outputChannel * filterHeight * filterWidth;
    int k = inputChannel;

    float alpha = 1;
    float beta  = 1;

    CUBLAS_CHECK(hipblasSgemm(device->cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_T, k, n, m, &alpha, x, k, dyMat, n, &beta, dw, k));

    device->free(dyMat);
}

template <>
void DeConv2d<double>::backwardGPUFilterImpl(Device *d, const double *x, double *dw, const double *dy,
                           int batch, int inputHeight, int inputWidth, int inputChannel,
                           int outputHeight, int outputWidth, int outputChannel,
                           int filterHeight, int filterWidth, int forwardStrideY, int forwardStrideX,
                           int padTop, int padLeft) {
    auto device = (GPUDevice*)d;
    int size = batch * inputHeight * inputWidth * outputChannel * filterHeight * filterWidth;

    auto dyMat = (double*)device->malloc(sizeof(double) * size);

    int minGrideSize;
    int blockSize;
    int grideSize;

    CUDA_CHECK(hipOccupancyMaxPotentialBlockSize(&minGrideSize, &blockSize, DeConv2dBackwardKernel<double>, 0, size));

    grideSize = (size + blockSize - 1) / blockSize;

    DeConv2dBackwardKernel<double> << <grideSize, blockSize >> > (dyMat, dy,
            batch, inputHeight, inputWidth, inputChannel,
            filterHeight, filterWidth,
            outputHeight, outputWidth, outputChannel,
            forwardStrideY, forwardStrideX,
            padTop, padLeft, size);

    int m = batch * inputHeight * inputWidth;
    int n = outputChannel * filterHeight * filterWidth;
    int k = inputChannel;

    double alpha = 1;
    double beta = 1;

    CUBLAS_CHECK(hipblasDgemm(device->cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_T, k, n, m, &alpha, x, k, dyMat, n, &beta, dw, k));

    device->free(dyMat);
}

#ifdef HAVE_HALF
template <>
void DeConv2d<half>::backwardGPUFilterImpl(Device *d, const half *x, half *dw, const half *dy,
                                    int batch, int inputHeight, int inputWidth, int inputChannel,
                                    int outputHeight, int outputWidth, int outputChannel,
                                    int filterHeight, int filterWidth, int forwardStrideY, int forwardStrideX,
                                    int padTop, int padLeft) {
    auto device = (GPUDevice*)d;
    int size = batch * inputHeight * inputWidth * outputChannel * filterHeight * filterWidth;

    auto dyMat = (half*)device->malloc(sizeof(half) * size);

    int blockSize = 1024;
    int grideSize = (size + blockSize - 1) / blockSize;

    DeConv2dBackwardKernel<half> << <grideSize, blockSize >> > (dyMat, dy,
        batch, inputHeight, inputWidth, inputChannel,
        filterHeight, filterWidth,
        outputHeight, outputWidth, outputChannel,
        forwardStrideY, forwardStrideX,
        padTop, padLeft, size);

    int m = batch * inputHeight * inputWidth;
    int n = outputChannel * filterHeight * filterWidth;
    int k = inputChannel;

    half alpha = 1;
    half beta = 1;

    CUBLAS_CHECK(hipblasHgemm(device->cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_T, k, n, m, &alpha, x, k, dyMat, n, &beta, dw, k));

    device->free(dyMat);
}
#endif // HAVE_HALF

template <typename T>
void DeConv2d<T>::backwardGPU(const std::vector<const Tensor<T>*> &inputs,
                 const Tensor<T> *output,
                 const Tensor<T> *outputGradient,
                 size_t index,
                 Tensor<T> *iGradient) {
    DEEP8_ARGUMENT_CHECK(0 == index || 1 == index, "the index is error");

    auto device = static_cast<GPUDevice*>(iGradient->device());

    auto xShape = inputs[0]->shape;
    auto wShape = inputs[1]->shape;
    auto yShape = output->shape;

    auto batch        = (int)xShape.dim(0);
    auto inputHeight  = (int)xShape.dim(1);
    auto inputWidth   = (int)xShape.dim(2);
    auto inputChannel = (int)xShape.dim(3);

    auto outputHeight  = (int)yShape.dim(1);
    auto outputWidth   = (int)yShape.dim(2);
    auto outputChannel = (int)yShape.dim(3);

    auto filterHeight = (int)wShape.dim(1);
    auto filterWidth  = (int)wShape.dim(2);

    auto padTop  = -(std::max<int>(0, outputHeight + filterHeight - (inputHeight - 1) * (int)(forwardStrideY) - 2) / 2);
    auto padLeft = -(std::max<int>(0, outputWidth  + filterWidth  - (inputWidth  - 1) * (int)(forwardStrideX) - 2) / 2);

    if (0 == index) {
        backwardGPUInputImpl(device, iGradient->data(), inputs[1]->data(), outputGradient->data(),
        batch, inputHeight, inputWidth, inputChannel,
		outputHeight, outputWidth, outputChannel,
		filterHeight, filterWidth, (int)forwardStrideY, (int)forwardStrideX,
		padTop, padLeft);
    } else if (1 == index) {
        backwardGPUFilterImpl(device, inputs[0]->data(), iGradient->data(), outputGradient->data(),
        batch, inputHeight, inputWidth, inputChannel,
		outputHeight, outputWidth, outputChannel,
		filterHeight, filterWidth, (int)forwardStrideY, (int)forwardStrideX,
		padTop, padLeft);
    }
}

DEEP8_DECLARATION_GPU_FUNC(DeConv2d);

#endif

}